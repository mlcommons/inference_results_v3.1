/*
 * Copyright (c) 2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>

#include "decoderPlugin.h"

#define CHECK(status)                                                                                                  \
    do                                                                                                                 \
    {                                                                                                                  \
        auto ret = (status);                                                                                           \
        if (ret != 0)                                                                                                  \
        {                                                                                                              \
            std::cout << "Cuda failure: " << ret << std::endl;                                                         \
            abort();                                                                                                   \
        }                                                                                                              \
    } while (0)

using namespace nvinfer1;
using nvinfer1::plugin::RNNTDecoderPlugin;
using nvinfer1::plugin::RNNTDecoderPluginCreator;

REGISTER_TENSORRT_PLUGIN(RNNTDecoderPluginCreator);

RNNTDecoderPlugin::RNNTDecoderPlugin(const PluginFieldCollection *fc) {
    int idx = 0;
    
    mNumLayers = *(int*)(fc->fields[idx].data);
    idx++;
    
    mHiddenSize = *(int*)(fc->fields[idx].data);
    idx++;
    
    mInputSize = *(int*)(fc->fields[idx].data);
    idx++;
    
    mDataType = *(nvinfer1::DataType*)(fc->fields[idx].data);
    idx++;
    
    mWeights_h = (void**)malloc(mNumLayers * sizeof(void*));
    
    for (int i = 0; i < mNumLayers; i++) {        
        mWeights_h[i] = (void*)fc->fields[idx].data;
        idx++;
    }
    
    mBias_h = (void**)malloc(mNumLayers * sizeof(void*));
    for (int i = 0; i < mNumLayers; i++) {        
        mBias_h[i] = (void*)fc->fields[idx].data;
        idx++;
    }
}

RNNTDecoderPlugin::RNNTDecoderPlugin(const void* data, size_t length) {
    const char *d = static_cast<const char*>(data);
    // Use maybe_unused attribute when updating to CUDA_STANDARD C++17
    #ifndef NDEBUG
    auto d_start = d;
    #endif
    read<int>(d, mNumLayers);
    read<int>(d, mHiddenSize);
    read<int>(d, mInputSize);
    
    read<nvinfer1::DataType>(d, mDataType);
    
    mWeights_h = (void**)malloc(mNumLayers * sizeof(void*));
    for (int i = 0; i < mNumLayers; i++) {        
        size_t dataTypeSize = 0;
        dataTypeSize = sizeof(half);
        
        size_t sz = 4 * mHiddenSize * ((i == 0 ? mInputSize : mHiddenSize) + mHiddenSize) * dataTypeSize;
               

        mWeights_h[i] = malloc(sz);
        memcpy(mWeights_h[i], d, sz);
        d += sz;
    }
    
    mBias_h = (void**)malloc(mNumLayers * sizeof(void*));
    for (int i = 0; i < mNumLayers; i++) {        
        size_t dataTypeSize = 0;
        dataTypeSize = sizeof(half);
        
        size_t sz = 8 * mHiddenSize * dataTypeSize;

        mBias_h[i] = malloc(sz);
        memcpy(mBias_h[i], d, sz);
        d += sz;
    }

    assert(d == d_start + length);
}

const char* RNNTDecoderPlugin::getPluginType() const noexcept
{
    return "RNNTDecoderPlugin";
}

const char* RNNTDecoderPlugin::getPluginVersion() const noexcept
{
    return "1";
}

void RNNTDecoderPlugin::setPluginNamespace(const char* libNamespace) noexcept
{
    mNamespace = libNamespace;
}

const char* RNNTDecoderPlugin::getPluginNamespace() const noexcept
{
    return mNamespace.c_str();
}

void RNNTDecoderPlugin::destroy() noexcept
{
    if (mWeights_h)
    {
        free(mWeights_h);
        mWeights_h = nullptr;
    }
    if (mBias_h) {
        free(mBias_h);
        mBias_h = nullptr;
    }
    delete this;
}

void RNNTDecoderPlugin::setCUDAInfo(hipStream_t mStreamh, hipblasHandle_t mCublas, void **mWeights_d, void **mBias_d, void *mWorkSpace_d) {
    this->mStreamh = mStreamh;
    this->mCublas = mCublas;
    this->mWeights_d = mWeights_d;
    this->mBias_d = mBias_d;
    this->mWorkSpace_d = mWorkSpace_d;
}

IPluginV2DynamicExt* RNNTDecoderPlugin::clone() const noexcept
{
    size_t sz = getSerializationSize();

    char* buff = (char*) malloc(getSerializationSize());

    serialize(buff);
   
    RNNTDecoderPlugin* ret = new RNNTDecoderPlugin(buff, sz);
    
    ret->setCUDAInfo(mStreamh, mCublas, mWeights_d, mBias_d, mWorkSpace_d);
    
    free(buff);

    return ret;
}

int RNNTDecoderPlugin::getNbOutputs() const noexcept
{
    return 3;
}

DimsExprs RNNTDecoderPlugin::getOutputDimensions(
    int32_t outputIndex, const DimsExprs* inputs, int32_t nbInputs, IExprBuilder& exprBuilder) noexcept
{

    assert(outputIndex >= 0 && outputIndex < this->getNbOutputs());

    return inputs[outputIndex];
}

bool RNNTDecoderPlugin::supportsFormatCombination(
    int32_t pos, const PluginTensorDesc* inOut, int32_t nbInputs, int32_t nbOutputs) noexcept
{
    if (inOut[pos].format != TensorFormat::kLINEAR)
        return false;

    // fp16 I/O
    if (mDataType == nvinfer1::DataType::kHALF) {
        bool allHalf = true;

        // Don't care about pos. If all are half pass it.
        // The way this is called doesn't fill all of inOut, it only fills it up to pos.
        for (int i = 0; i <= pos; i++) {
            if (inOut[i].type != DataType::kHALF) {
                allHalf = false;
            }
        }
        
        if (allHalf) {
            return true;
        }
        return false;
    }
    return false;
}

void RNNTDecoderPlugin::configurePlugin(
    const DynamicPluginTensorDesc* in, int32_t nbInputs, const DynamicPluginTensorDesc* out, int32_t nbOutputs) noexcept
{
    // mInputSize = in[0].desc.dims.d[in[0].desc.dims.nbDims - 1];
}

int RNNTDecoderPlugin::initialize() noexcept
{
    if (!mInitialized)
    {
        CHECK(hipblasCreate(&mCublas));

        CHECK(hipblasSetMathMode(mCublas, HIPBLAS_TENSOR_OP_MATH));
        
        CHECK(hipStreamCreate(&mStreamh));
            
        
        mWeights_d = (void**)malloc(mNumLayers * sizeof(void*));
        
        for (int i = 0; i < mNumLayers; i++) {        
            size_t dataTypeSize = 0;
            if (mDataType == DataType::kHALF) {
                dataTypeSize = sizeof(half);
            }
            
            size_t sz = 4 * mHiddenSize * ((i == 0 ? mInputSize : mHiddenSize) + mHiddenSize) * dataTypeSize;
            
            CHECK(hipMalloc(&mWeights_d[i], sz));
    
            CHECK(hipMemcpy(mWeights_d[i], mWeights_h[i], sz, hipMemcpyHostToDevice));        
        }
        
        mBias_d = (void**)malloc(mNumLayers * sizeof(void*));
        
        for (int i = 0; i < mNumLayers; i++) {        
            size_t dataTypeSize = 0;
            if (mDataType == DataType::kHALF) {
                dataTypeSize = sizeof(half);
            }
            
            size_t sz = 8 * mHiddenSize * dataTypeSize;
            CHECK(hipMalloc(&mBias_d[i], sz));
            
            CHECK(hipMemcpy(mBias_d[i], mBias_h[i], sz, hipMemcpyHostToDevice));        
           
        }        
        
        
        mWorkSpace_d = NULL;// CHECK(hipMalloc(&mWorkSpace_d, getWorkspaceSize()));
    }

    return hipSuccess;
}

void RNNTDecoderPlugin::terminate() noexcept
{
    if (mCublas)
    {
        CHECK(hipblasDestroy(mCublas));
        mCublas = nullptr;
    }
    
    if (mStreamh) {
        CHECK(hipStreamDestroy(mStreamh));
        mStreamh = nullptr;
    }
            
    if (mWeights_d) {
        for (int i = 0; i < mNumLayers; i++) {           
            if (mWeights_d[i]) {                
                hipFree(mWeights_d[i]);
                mWeights_d[i] = nullptr;
            }
        }
        free(mWeights_d);
        mWeights_d = nullptr;
    }
    
    if (mBias_d) {
        for (int i = 0; i < mNumLayers; i++) {           
            if (mBias_d[i]) {                
                hipFree(mBias_d[i]);
                mBias_d[i] = nullptr;
            }
        }
        free(mBias_d);
        mBias_d = nullptr;
    }
    
    if (!mWorkSpace_d) {
        hipFree(mWorkSpace_d);
        mWorkSpace_d = nullptr;
    }
}

size_t RNNTDecoderPlugin::getWorkspaceSize(
    const PluginTensorDesc* inputs, int nbInputs, const PluginTensorDesc* outputs, int nbOutputs) const noexcept
{
    size_t size = 0;

    int batchSize = inputs[0].dims.d[0];

    // tmp_io
    size += mNumLayers * mHiddenSize * batchSize * sizeof(half);

    // tmp_i
    size += mHiddenSize * batchSize * 4 * sizeof(half);

    // tmp_h
    size += mNumLayers * mHiddenSize * batchSize * 4 * sizeof(half);

    return size;
}

int RNNTDecoderPlugin::enqueue(const PluginTensorDesc* inputDesc, const PluginTensorDesc* outputDesc,
    const void* const* inputs, void* const* outputs, void* workspace, hipStream_t stream) noexcept
{
    int batchSize = inputDesc[0].dims.d[0];

    int effectiveBatch = batchSize;

    void *tmp_io = NULL;
    void *tmp_i = NULL; 
    void *tmp_h = NULL; 
    
    tmp_io = workspace;
    tmp_i = (void*)((char*)(tmp_io) + mNumLayers * mHiddenSize * effectiveBatch * sizeof(half));
    tmp_h = (void*)((char*)(tmp_i) + mHiddenSize * effectiveBatch * 4 * sizeof(half));
    
    hipEvent_t event;
    CHECK(hipEventCreate(&event, hipEventDisableTiming));
    CHECK(hipEventRecord(event, stream));  
    CHECK(hipStreamWaitEvent(mStreamh, event, 0));
    CHECK(hipEventDestroy(event));
   
    if (mDataType == nvinfer1::DataType::kHALF) {
        decoderStep<half, HIP_R_16F, half, HIP_R_16F, half>
                (mHiddenSize, 
                 mInputSize,
                 effectiveBatch, 
                 1,
                 mNumLayers,
                 this->mCublas,
                 (half*)inputs[0], // x 
                 (half*)inputs[1], // hx, 
                 (half*)inputs[2], // cx, 
                 (half**)mWeights_d,
                 (half**)mBias_d, // bias
                 (half*)outputs[0], // y, 
                 (half*)outputs[1], // hy, 
                 (half*)outputs[2], // cy,
                 (half*)tmp_io,
                 (half*)tmp_i,
                 (half*)tmp_h,
                 stream,
                 mStreamh);
    }

    return 0;
}

size_t RNNTDecoderPlugin::getSerializationSize() const noexcept
{
    size_t sz = sizeof(mNumLayers) + sizeof(mHiddenSize) + sizeof(mInputSize) + sizeof(mDataType);

    // Weights
    for (int i = 0; i < mNumLayers; i++) {
        size_t dataTypeSize = 0;
        if (mDataType == DataType::kHALF) {
            dataTypeSize = sizeof(half);
        }
       
        sz += 4 * mHiddenSize * ((i == 0 ? mInputSize : mHiddenSize) + mHiddenSize) * dataTypeSize;
    }
    
    // Bias
    for (int i = 0; i < mNumLayers; i++) {
        size_t dataTypeSize = 0;
        if (mDataType == DataType::kHALF) {
            dataTypeSize = sizeof(half);
        }
       
        sz += 8 * mHiddenSize * dataTypeSize;
    }

    return sz;
}

void RNNTDecoderPlugin::serialize(void* buffer) const noexcept
{
    char* d = static_cast<char*>(buffer);
// Use maybe_unused attribute when updating to CUDA_STANDARD C++17
#ifndef NDEBUG
    auto d_start = d;
    #endif
    
    write<int>(d, mNumLayers);
    write<int>(d, mHiddenSize);        
    write<int>(d, mInputSize);
    write<nvinfer1::DataType>(d, mDataType);
    
    
    for (int i = 0; i < mNumLayers; i++) {        
        size_t dataTypeSize = 0;
        if (mDataType == DataType::kHALF) {
            dataTypeSize = sizeof(half);
        }
        
        size_t sz = 4 * mHiddenSize * ((i == 0 ? mInputSize : mHiddenSize) + mHiddenSize) * dataTypeSize;

        memcpy(d, mWeights_h[i], sz);
        d += sz;
    }

    for (int i = 0; i < mNumLayers; i++) {        
        size_t dataTypeSize = 0;
        if (mDataType == DataType::kHALF) {
            dataTypeSize = sizeof(half);
        }
        
        size_t sz = 8 * mHiddenSize * dataTypeSize;

        memcpy(d, mBias_h[i], sz);
        d += sz;
    }

    assert(d == d_start + getSerializationSize());
}

nvinfer1::DataType RNNTDecoderPlugin::getOutputDataType(
    int index, const nvinfer1::DataType* inputTypes, int nbInputs) const noexcept
{
    return mDataType;
}

template <typename T>
void RNNTDecoderPlugin::write(char*& buffer, const T& val) const
{
    *reinterpret_cast<T*>(buffer) = val;
    buffer += sizeof(T);
}

template <typename T>
void RNNTDecoderPlugin::read(const char*& buffer, T& val) const
{
    val = *reinterpret_cast<const T*>(buffer);
    buffer += sizeof(T);
}

const char* RNNTDecoderPluginCreator::getPluginName() const noexcept
{
    return "RNNTDecoderPlugin";
}

const char* RNNTDecoderPluginCreator::getPluginVersion() const noexcept
{
    return "1";
}

const PluginFieldCollection* RNNTDecoderPluginCreator::getFieldNames() noexcept
{
    return nullptr;
}

void RNNTDecoderPluginCreator::setPluginNamespace(const char* libNamespace) noexcept
{
    mNamespace = libNamespace;
}

const char* RNNTDecoderPluginCreator::getPluginNamespace() const noexcept
{
    return mNamespace.c_str();
}

IPluginV2DynamicExt* RNNTDecoderPluginCreator::createPlugin(const char* name, const PluginFieldCollection* fc) noexcept
{
    return new RNNTDecoderPlugin(fc);
}

IPluginV2DynamicExt* RNNTDecoderPluginCreator::deserializePlugin(
    const char* name, const void* serialData, size_t serialLength) noexcept
{
    return new RNNTDecoderPlugin(serialData, serialLength);
}
