#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "dlrmHelper.h"
#include "dlrmInteractionsPluginKernel.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>

#define gpuErrChk(ans)                                                                                                 \
    {                                                                                                                  \
        gpuAssert((ans), __FILE__, __LINE__);                                                                          \
    }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

using namespace std;

__global__ void decompression_kernel_opt(
    const int* const __restrict__ com_data_gpu, int* const __restrict__ decom_data_gpu, const int num_ui_pairs)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    const int total_threads = blockDim.x * gridDim.x;
    const int max_per_thread = (num_ui_pairs + total_threads - 1) / total_threads;
    const int odd_offset = index > total_threads / 2 ? 1 : 0;
    index = index - total_threads / 2 * odd_offset;

#pragma unroll
    for (int i = 0; i < max_per_thread; i++)
    {
        // first half of threads work on even ui_pairs
        // second half of threds work on odd ui_pairs
        // this reduces warp divergence since adjacent threads execute
        // same block of switch statement
        long int ui_pair_idx = index * 2 + odd_offset + (total_threads * i);

        if (ui_pair_idx < num_ui_pairs)
        {

            int4 inp1, inp2, inp3, inp4;
            int4 outp1, outp2, outp3, outp4, outp5, outp6, outp7;

            // Reading the values from DRAM in 128 bit packets
            inp1 = *(int4*) &com_data_gpu[(ui_pair_idx * 16) + 0];
            inp2 = *(int4*) &com_data_gpu[(ui_pair_idx * 16) + 4];
            inp3 = *(int4*) &com_data_gpu[(ui_pair_idx * 16) + 8];
            inp4 = *(int4*) &com_data_gpu[(ui_pair_idx * 16) + 12];

            // Writing the values into DRAM in 128 bit packets
            int4* p_decom_data_gpu = NULL;

            // Every other ui_pair_idx is 16 byte aligned
            // 2 cases:
            //  1. when ui_pair_idx is 16 byte aligned, write int4s starting at base address
            //  2. when ui_pair_idx is not 16 byte aligned, write int4s starting at base address + 8 bytes
            switch (((ui_pair_idx * 26) % 4))
            {
            case 0:
                p_decom_data_gpu = (int4*) &decom_data_gpu[(ui_pair_idx * 26) + 0];
                outp1.x = inp1.x;
                outp1.y = (inp1.y & 0xFFFF'0000) >> 16;
                outp1.z = inp1.y & 0xFFFF;
                outp1.w = (inp1.z & 0xFFFF'0000) >> 16;
                p_decom_data_gpu[0] = outp1;

                outp2.x = inp1.z & 0xFFFF;
                outp2.y = (inp3.x & 0xFF00'0000) >> 24;
                outp2.z = (inp1.w & 0xFFFF'0000) >> 16;
                outp2.w = inp1.w & 0xFFFF;
                p_decom_data_gpu[1] = outp2;

                outp3.x = (inp3.x & 0xFF'0000) >> 16;
                outp3.y = inp2.x;
                outp3.z = inp2.y;
                outp3.w = inp2.z;
                p_decom_data_gpu[2] = outp3;

                outp4.x = (inp3.x & 0xFF00) >> 8;
                outp4.y = (inp2.w & 0xFFFF'0000) >> 16;
                outp4.z = inp2.w & 0xFFFF;
                outp4.w = inp3.x & 0xFF;
                p_decom_data_gpu[3] = outp4;

                outp5.x = (inp3.y & 0xFF00'0000) >> 24;
                outp5.y = (inp3.y & 0xFF'FF00) >> 8;
                outp5.z = inp3.y & 0xFF;
                outp5.w = inp3.z;
                p_decom_data_gpu[4] = outp5;

                outp6.x = inp3.w;
                outp6.y = inp4.x;
                outp6.z = inp4.y;
                outp6.w = (inp4.z & 0xFFFF'0000) >> 16;
                p_decom_data_gpu[5] = outp6;

                outp7.x = (inp4.z & 0xFF00) >> 8;
                outp7.y = inp4.z & 0xFF;
                decom_data_gpu[(ui_pair_idx * 26) + 24] = outp7.x;
                decom_data_gpu[(ui_pair_idx * 26) + 25] = outp7.y;
                break;
            case 1: break;

            case 2:
                p_decom_data_gpu = (int4*) &decom_data_gpu[(ui_pair_idx * 26) + 2];
                outp1.x = inp1.x;
                outp1.y = (inp1.y & 0xFFFF'0000) >> 16;
                decom_data_gpu[(ui_pair_idx * 26) + 0] = outp1.x;
                decom_data_gpu[(ui_pair_idx * 26) + 1] = outp1.y;

                outp2.x = inp1.y & 0xFFFF;
                outp2.y = (inp1.z & 0xFFFF'0000) >> 16;
                outp2.z = inp1.z & 0xFFFF;
                outp2.w = (inp3.x & 0xFF00'0000) >> 24;
                p_decom_data_gpu[0] = outp2;

                outp3.x = (inp1.w & 0xFFFF'0000) >> 16;
                outp3.y = inp1.w & 0xFFFF;
                outp3.z = (inp3.x & 0xFF'0000) >> 16;
                outp3.w = inp2.x;
                p_decom_data_gpu[1] = outp3;

                outp4.x = inp2.y;
                outp4.y = inp2.z;
                outp4.z = (inp3.x & 0xFF00) >> 8;
                outp4.w = (inp2.w & 0xFFFF'0000) >> 16;
                p_decom_data_gpu[2] = outp4;

                outp5.x = inp2.w & 0xFFFF;
                outp5.y = inp3.x & 0xFF;
                outp5.z = (inp3.y & 0xFF00'0000) >> 24;
                outp5.w = (inp3.y & 0xFF'FF00) >> 8;
                p_decom_data_gpu[3] = outp5;

                outp6.x = inp3.y & 0xFF;
                outp6.y = inp3.z;
                outp6.z = inp3.w;
                outp6.w = inp4.x;
                p_decom_data_gpu[4] = outp6;

                outp7.x = inp4.y;
                outp7.y = (inp4.z & 0xFFFF'0000) >> 16;
                outp7.z = (inp4.z & 0xFF00) >> 8;
                outp7.w = inp4.z & 0xFF;
                p_decom_data_gpu[5] = outp7;
                break;
            case 3: break;
            }
        }
    }
}

void run_decompression(int* com_data, int* decom_data, int num_ui_pairs, hipStream_t stream)
{
    // TODO: how should launch config change for different architectures
    decompression_kernel_opt<<<128, 1024, 0, stream>>>(com_data, decom_data, num_ui_pairs);
}
