#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hipcub/hipcub.hpp>
#include <vector>

#include "ssdOpt.h"
#include "ssdOptMacros.h"
#include "topK.h"

#include "nms_common.h"

//#undef USE_CUB_SEGMENTED_SORT

template <typename KeyT, typename ValueT>
size_t cubSortPairsWorkspaceSize(int num_items, int num_segments)
{
    size_t temp_storage_bytes = 0;
    hipcub::DeviceSegmentedRadixSort::SortPairsDescending((void*) NULL, temp_storage_bytes, (const KeyT*) NULL,
        (KeyT*) NULL, (const ValueT*) NULL, (ValueT*) NULL,
        num_items,    // # items
        num_segments, // # segments
        (const int*) NULL, (const int*) NULL);
    return temp_storage_bytes;
}

namespace nvinfer1
{
namespace plugin
{

namespace
{

#if USE_CUB_SEGMENTED_SORT == 1

__global__ void get_cub_offsets_kernel(int* begin_offset, int* end_offset, int num, int stride)
{

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= num)
        return;
    int num_items = max(begin_offset[idx], 0);
    end_offset[idx] = idx * stride + num_items;
    begin_offset[idx] = idx * stride;
}

template <typename T_SCORE>
__global__ void top_k_score_per_image_prepare_outputs(T_SCORE* input_scores, int* input_indices, T_SCORE* output_scores,
    int* output_indices, int* in_end_offsets, int* out_active_counts, int items, int segments, int num_top_k)
{
    int segment_id = blockIdx.y;
    const int stride = items / segments;
    int elem_id = blockIdx.x * blockDim.x + threadIdx.x;

    if (segment_id >= segments)
        return;
    int num_items = min(max(0, in_end_offsets[segment_id] - segment_id * stride), num_top_k);
    out_active_counts[segment_id] = num_items;

    if (elem_id >= num_top_k)
        return;

    int idx = elem_id + stride * segment_id;

    // make things consistent with the reference for now (pad up to top_k with 0/-1)
    output_scores[idx] = (elem_id < num_items) ? input_scores[idx] : 0.0F;
    output_indices[idx] = (elem_id < num_items) ? input_indices[idx] : -1;

    // output_scores[idx] = input_scores[idx];
    // output_indices[idx] = input_indices[idx];
}

template <int BLOCK_THREADS>
__global__ void get_active_counts(int* in_end_offsets, int* out_active_counts, int items, int segments, int num_top_k)
{

    int segment_id = blockIdx.x * BLOCK_THREADS + threadIdx.x;

    const int stride = items / segments;

    if (segment_id >= segments)
        return;

    out_active_counts[segment_id] = min(max(0, in_end_offsets[segment_id] - segment_id * stride), num_top_k);

    // printf("%d, %d\n", segment_id, min(max(0, in_end_offsets[segment_id] - segment_id * stride), num_top_k));
}

#endif

// sort one segment per cta
template <typename T_SCORE, int BLOCK_THREADS, int ELEMENTS_PER_THREAD>
__global__ void blockSortKernel(const T_SCORE* d_keys_in, T_SCORE* d_keys_out, const int32_t* d_values_in,
    int32_t* d_values_out, const int32_t* active_counts, int num_items, int stride_items, int num_segments)
{
    // Specialize BlockRadixSort for a 1D block
    typedef hipcub::BlockRadixSort<T_SCORE, BLOCK_THREADS, ELEMENTS_PER_THREAD, int32_t> BlockRadixSort;

    // Allocate shared memory for BlockRadixSort
    __shared__ typename BlockRadixSort::TempStorage temp_storage;

    if (blockIdx.x >= num_segments)
        return;

    int num_active_items = active_counts[blockIdx.x];

    // Obtain a segment of consecutive items that are blocked across threads
    T_SCORE thread_keys[ELEMENTS_PER_THREAD];
    int32_t thread_values[ELEMENTS_PER_THREAD];

    int32_t block_offset = blockIdx.x * stride_items;
    hipcub::LoadDirectStriped<BLOCK_THREADS>(threadIdx.x, d_keys_out + block_offset, thread_keys, num_active_items, 0);
    hipcub::LoadDirectStriped<BLOCK_THREADS>(
        threadIdx.x, d_values_out + block_offset, thread_values, num_active_items, -1);
    __syncthreads();

    // Collectively sort the keys and values among block threads
    BlockRadixSort(temp_storage).SortDescendingBlockedToStriped(thread_keys, thread_values);

    // Store output in striped fashion
    hipcub::StoreDirectStriped<BLOCK_THREADS>(threadIdx.x, d_keys_out + block_offset, thread_keys, num_items);
    hipcub::StoreDirectStriped<BLOCK_THREADS>(threadIdx.x, d_values_out + block_offset, thread_values, num_items);
}

/// block sort kernel
template <typename T_SCORE>
void blockSort(const T_SCORE* d_keys_in, T_SCORE* d_keys_out, const int32_t* d_values_in, int32_t* d_values_out,
    const int32_t* active_counts, int num_items, int stride_items, int num_segments, hipStream_t stream)
{
    if (num_items == 0)
        return;

    int kernel_index = div_up(num_items, 128) - 1;
    int warps_per_cta = (kernel_index + 1) * 128 / 32;
    assert(warps_per_cta <= 32);

    dim3 block(warps_per_cta * 32);
    dim3 grid(num_segments);

    using kernel_func = void (*)(const T_SCORE* d_keys_in, T_SCORE* d_keys_out, const int32_t* d_values_in,
        int32_t* d_values_out, const int32_t* active_counts, int num_items, int stride_items, int num_segments);

    static const kernel_func kernel_funcs[] = {
        &blockSortKernel<T_SCORE, 128, 1>,
        &blockSortKernel<T_SCORE, 256, 1>,
        &blockSortKernel<T_SCORE, 384, 1>,
        &blockSortKernel<T_SCORE, 512, 1>,
        &blockSortKernel<T_SCORE, 640, 1>,
        &blockSortKernel<T_SCORE, 768, 1>,
        &blockSortKernel<T_SCORE, 896, 1>,
        &blockSortKernel<T_SCORE, 1024, 1>,
    };
    kernel_funcs[kernel_index]<<<grid, block, 0, stream>>>(
        d_keys_in, d_keys_out, d_values_in, d_values_out, active_counts, num_items, stride_items, num_segments);
}

} // namespace

template <typename T_SCORE>
ssdStatus_t topKScoresPerImage_gpu(hipStream_t stream, const int num_images, const int num_items_per_image,
    const int num_top_k, void* unsorted_scores, void* unsorted_bbox_indices, void* sorted_scores,
    void* sorted_bbox_indices, void* active_count, void* active_count_per_batch, void* temp_active_count,
    size_t temp_storage_bytes, void* workspace)
{
    void* d_offsets = workspace;
    void* cubWorkspace = nextWorkspacePtr((int8_t*) d_offsets, (num_images + 1) * sizeof(int));

    const int num_classes = num_items_per_image / num_top_k;

#if USE_CUB_SEGMENTED_SORT == 1

    segmented_scan(
        (int*) active_count, (int*) active_count, (int*) active_count_per_batch, num_images, num_classes, stream);

    compact_segments(unsorted_scores, unsorted_bbox_indices, sorted_scores, sorted_bbox_indices, (int*) active_count,
        (int*) active_count_per_batch, num_classes, num_images, num_items_per_image, num_items_per_image, stream);

    // get offsets
    get_cub_offsets_kernel<<<div_up(num_images, 128), 128, 0, stream>>>(
        (int*) active_count_per_batch, (int*) temp_active_count, num_images, num_items_per_image);

    int items = num_images * num_items_per_image;

    hipcub::DeviceSegmentedRadixSort::SortPairsDescending(workspace, temp_storage_bytes,
        reinterpret_cast<T_SCORE*>(sorted_scores), // keys_in
        reinterpret_cast<T_SCORE*>(unsorted_scores),
        reinterpret_cast<int*>(sorted_bbox_indices), // indicies_in
        reinterpret_cast<int*>(unsorted_bbox_indices), items, num_images,
        //(int *)active_count,
        (int*) active_count_per_batch, (int*) temp_active_count, 0, sizeof(T_SCORE) * 8, stream);

    if (0)
    {
        // copy to output. write a smarter kernel based on the active counts
        hipMemcpyAsync(sorted_scores, unsorted_scores, items * sizeof(T_SCORE), hipMemcpyDefault, stream);
        hipMemcpyAsync(sorted_bbox_indices, unsorted_bbox_indices, items * sizeof(int), hipMemcpyDefault, stream);

        get_active_counts<128><<<div_up(num_images, 128), 128, 0, stream>>>(reinterpret_cast<int*>(temp_active_count),
            reinterpret_cast<int*>(active_count_per_batch), items, num_images, num_top_k);
    }
    else
    {

        top_k_score_per_image_prepare_outputs<<<dim3(div_up(num_top_k, 128), num_images, 1), 128, 0, stream>>>(
            reinterpret_cast<T_SCORE*>(unsorted_scores), reinterpret_cast<int*>(unsorted_bbox_indices),
            reinterpret_cast<T_SCORE*>(sorted_scores), reinterpret_cast<int*>(sorted_bbox_indices),
            reinterpret_cast<int*>(temp_active_count), reinterpret_cast<int*>(active_count_per_batch), items,
            num_images, num_top_k);
    }

#else

    uint32_t num_warps = (num_items_per_image > 1024) ? 32 : (num_items_per_image + 31) / 32;

    // const int WARP_SZ = 32;
    const int BLOCK_THREADS = 512;

    // printf("top_k Per Image\n");

    top_k_multi_pass((int*) (unsorted_scores), (int*) unsorted_bbox_indices, (int*) (sorted_scores),
        (int*) sorted_bbox_indices, (int*) active_count, (int*) active_count_per_batch, num_items_per_image,
        num_items_per_image, num_top_k, num_classes, num_images, stream);

    void* block_sort_scores = unsorted_scores;
    void* block_sort_indices = unsorted_bbox_indices;

    // dim3 block(num_warps * WARP_SZ);
    // dim3 grid(num_images);
    // block.x = num_warps * 32;

    blockSort<T_SCORE>((const T_SCORE*) (block_sort_scores), (T_SCORE*) (sorted_scores),
        (const int*) (block_sort_indices), (int*) (sorted_bbox_indices), (int*) active_count_per_batch, num_top_k,
        num_items_per_image, num_images, stream);

#endif

    CSC(hipGetLastError(), STATUS_FAILURE);
    return STATUS_SUCCESS;
}

// sortScoresPerImage LAUNCH CONFIG {{{
typedef ssdStatus_t (*tkspiFunc)(
    hipStream_t, const int, const int, const int, void*, void*, void*, void*, void*, void*, void*, size_t, void*);
struct tkspiLaunchConfig
{
    DType_t t_score;
    tkspiFunc function;

    tkspiLaunchConfig(DType_t t_score)
        : t_score(t_score)
    {
    }
    tkspiLaunchConfig(DType_t t_score, tkspiFunc function)
        : t_score(t_score)
        , function(function)
    {
    }
    bool operator==(const tkspiLaunchConfig& other)
    {
        return t_score == other.t_score;
    }
};

using nvinfer1::DataType;

static std::vector<tkspiLaunchConfig> tkspiFuncVec;
bool tkspiInit()
{
    tkspiFuncVec.push_back(tkspiLaunchConfig(DataType::kFLOAT, topKScoresPerImage_gpu<float>));
    return true;
}

static bool initialized = tkspiInit();
//}}}

ssdStatus_t topKScoresPerImage(hipStream_t stream, const int num_images, const int num_items_per_image,
    const int num_top_k, const DType_t DT_SCORE, void* unsorted_scores, void* unsorted_bbox_indices,
    void* sorted_scores, void* sorted_bbox_indices, void* active_count, void* active_count_per_gpu,
    void* temp_active_count, size_t temp_storage_bytes, void* workspace)
{
    tkspiLaunchConfig lc = tkspiLaunchConfig(DT_SCORE);
    for (unsigned i = 0; i < tkspiFuncVec.size(); ++i)
    {
        if (lc == tkspiFuncVec[i])
        {
            DEBUG_PRINTF("topKScoresPerImage kernel %d\n", i);
            return tkspiFuncVec[i].function(stream, num_images, num_items_per_image, num_top_k, unsorted_scores,
                unsorted_bbox_indices, sorted_scores, sorted_bbox_indices, active_count, active_count_per_gpu,
                temp_active_count, temp_storage_bytes, workspace);
        }
    }
    return STATUS_BAD_PARAM;
}

size_t topKScoresPerImageWorkspaceSize(
    const int num_images, const int num_items_per_image, const int num_top_k, const DType_t DT_SCORE)
{
    const int arrayLen = num_images * num_items_per_image;
    size_t wss[2];
    wss[0] = (num_images + 1) * sizeof(int); // offsets
    if (DT_SCORE == DataType::kFLOAT)
    {
        wss[1] = cubSortPairsWorkspaceSize<float, int>(arrayLen, num_images); // cub workspace
    }
    else
    {
        printf("SCORE type not supported.\n");
        return (size_t) -1;
    }

    return calculateTotalWorkspaceSize(wss, 2);
}

} // namespace plugin
} // namespace nvinfer1
