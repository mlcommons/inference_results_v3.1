#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <fstream>
#include <vector>

#include <hipcub/hipcub.hpp>

#include "ssdOpt.h"
#include "ssdOptMacros.h"

template <typename T>
void saveDeviceBuffer(const T* buf, size_t count, std::ofstream& output)
{
    T* hBuf = new T[count];
    hipDeviceSynchronize();
    hipMemcpy(hBuf, buf, count * sizeof(T), hipMemcpyDeviceToHost);

    for (size_t i = 0; i < count; i += 4)
    {
        printf("%d ", i / 4);
        for (int j = 0; j < 4; j++)
        {
            printf("%f ", hBuf[i + j]);
        }
        printf("\n");
    }

    output.write((char*) hBuf, count * sizeof(T));
    delete[] hBuf;
}

namespace nvinfer1
{
namespace plugin
{

template <typename T_BBOX>
__device__ inline T_BBOX clip_bbox_coord(bool clip, T_BBOX* in)
{
    return (clip) ? max(min(*in, T_BBOX(1.)), T_BBOX(0.)) : *in;
}

template <typename T_BBOX, typename T_SCORE, unsigned nthds_per_cta>
__launch_bounds__(nthds_per_cta) __global__ void gatherTopDetectionsOpt_kernel(const bool shareLocation,
    const bool clip, const int numImages, const int numPredsPerClass, const int numClasses, const int topK,
    const int keepTopK, const int* indices, const T_SCORE* scores, const T_BBOX* bboxData, float* topDetections)
{
    typedef hipcub::BlockReduce<int, nthds_per_cta> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;

    assert(keepTopK <= topK);

    const int imgId = blockIdx.x;
    const int imgBase = imgId * (7 * keepTopK + 1);
    const int offset = imgId * numClasses * topK;
    const int bboxOffset = imgId * (shareLocation ? numPredsPerClass : (numClasses * numPredsPerClass));

    int isValid = 0;
    int aggregate = 0;

    int finish = ((keepTopK + nthds_per_cta - 1) / nthds_per_cta) * nthds_per_cta;
    // float* topDetections_float = reinterpret_cast<float*>(topDetections);
    const float* bboxData_float = reinterpret_cast<const float*>(bboxData);
    for (int detId = threadIdx.x; detId < finish; detId += nthds_per_cta)
    {
        if (detId < keepTopK)
        {
            const int index = indices[offset + detId];
            const T_SCORE score = scores[offset + detId];

            isValid = (index == -1) ? 0 : 1;

            const int bboxId
                = ((shareLocation ? (index % numPredsPerClass) : index % (numClasses * numPredsPerClass)) + bboxOffset)
                * 4;

#if SSD_RETINA_NET == 1
            topDetections[imgBase + detId * 7] = (isValid) ? score : 0; // image id
            // clipped bbox ymin
            topDetections[imgBase + detId * 7 + 1] = (isValid) ? clip_bbox_coord(clip, &bboxData_float[bboxId + 1]) : 0;
            // clipped bbox xmin
            topDetections[imgBase + detId * 7 + 2] = (isValid) ? clip_bbox_coord(clip, &bboxData_float[bboxId]) : 0;
            // clipped bbox ymax
            topDetections[imgBase + detId * 7 + 3] = (isValid) ? clip_bbox_coord(clip, &bboxData_float[bboxId + 3]) : 0;
            // clipped bbox xmax
            topDetections[imgBase + detId * 7 + 4] = (isValid) ? clip_bbox_coord(clip, &bboxData_float[bboxId + 2]) : 0;
            topDetections[imgBase + detId * 7 + 5] = (isValid) ? score : 0; // confidence score
            topDetections[imgBase + detId * 7 + 6]
                = (isValid) ? (index % (numClasses * numPredsPerClass)) / numPredsPerClass : -1; // label
#else
            topDetections[imgBase + detId * 7] = imgId; // image id
            // clipped bbox ymin
            topDetections[imgBase + detId * 7 + 1] = (isValid) ? clip_bbox_coord(clip, &bboxData[bboxId + 1]) : 0;
            // clipped bbox xmin
            topDetections[imgBase + detId * 7 + 2] = (isValid) ? clip_bbox_coord(clip, &bboxData[bboxId]) : 0;
            // clipped bbox ymax
            topDetections[imgBase + detId * 7 + 3] = (isValid) ? clip_bbox_coord(clip, &bboxData[bboxId + 3]) : 0;
            // clipped bbox xmax
            topDetections[imgBase + detId * 7 + 4] = (isValid) ? clip_bbox_coord(clip, &bboxData[bboxId + 2]) : 0;
            topDetections[imgBase + detId * 7 + 5] = (isValid) ? score : 0; // confidence score
            topDetections[imgBase + detId * 7 + 6]
                = (isValid) ? (index % (numClasses * numPredsPerClass)) / numPredsPerClass : -1; // label
#endif
        }
        else
        {
            isValid = 0;
        }

        aggregate += BlockReduce(temp_storage).Reduce(isValid, hipcub::Sum());
    }
    if (threadIdx.x == 0)
    {
        DEBUG_PRINTF("aggregate = %d, keepTopK = %d\n", aggregate, keepTopK);
#if SSD_RETINA_NET == 1
        topDetections[imgBase + 7 * keepTopK] = aggregate;
#else
        ((int*) topDetections)[imgBase + 7 * keepTopK] = aggregate;
#endif
    }
}

template <typename T_BBOX, typename T_SCORE>
ssdStatus_t gatherTopDetectionsOpt_gpu(hipStream_t stream, const bool shareLocation, const bool clip,
    const int numImages, const int numPredsPerClass, const int numClasses, const int topK, const int keepTopK,
    const void* indices, const void* scores, const void* bboxData, void* topDetections)
{

    static int iter = 0;
    if (iter == -1)
    {
        // debug

        auto output_file = std::ofstream("final_bboxes.bin", std::ios::binary);

        // std::vector<int> header = {num_top_k, num_preds_per_class, segments};
        // output_file.write((char *) &header[0], header.size() * sizeof(int));
        saveDeviceBuffer((const float*) bboxData, numImages * 4 * keepTopK, output_file);
        output_file.close();

        // exit(1);
    }
    else
    {
        iter++;
    };

    hipMemsetAsync(topDetections, 0, numImages * (7 * keepTopK + 1) * sizeof(float), stream);
    const int BS = 128;
    int GS = numImages;
    gatherTopDetectionsOpt_kernel<T_BBOX, T_SCORE, BS><<<GS, BS, 0, stream>>>(shareLocation, clip, numImages,
        numPredsPerClass, numClasses, topK, keepTopK, (int*) indices, (T_SCORE*) scores, (T_BBOX*) bboxData,
        /*(int*) keepCount,*/ (float*) topDetections);

    // CSC(hipGetLastError(), STATUS_FAILURE);
    return STATUS_SUCCESS;
}

// gatherTopDetectionsOpt LAUNCH CONFIG {{{
typedef ssdStatus_t (*gtdFunc)(hipStream_t, const bool, const bool, const int, const int, const int, const int,
    const int, const void*, const void*, const void*, void*);
struct gtdLaunchConfig
{
    DType_t t_bbox;
    DType_t t_score;
    gtdFunc function;

    gtdLaunchConfig(DType_t t_bbox, DType_t t_score)
        : t_bbox(t_bbox)
        , t_score(t_score)
    {
    }
    gtdLaunchConfig(DType_t t_bbox, DType_t t_score, gtdFunc function)
        : t_bbox(t_bbox)
        , t_score(t_score)
        , function(function)
    {
    }
    bool operator==(const gtdLaunchConfig& other)
    {
        return t_bbox == other.t_bbox && t_score == other.t_score;
    }
};

using nvinfer1::DataType;

static std::vector<gtdLaunchConfig> gtdFuncVec;

bool gtdOptInit()
{
    gtdFuncVec.push_back(gtdLaunchConfig(DataType::kFLOAT, DataType::kFLOAT, gatherTopDetectionsOpt_gpu<float, float>));
    return true;
}

static bool initialized = gtdOptInit();

//}}}

ssdStatus_t gatherTopDetectionsOpt(hipStream_t stream, const bool shareLocation, const bool clip, const int numImages,
    const int numPredsPerClass, const int numClasses, const int topK, const int keepTopK, const DType_t DT_BBOX,
    const DType_t DT_SCORE, const void* indices, const void* scores, const void* bboxData, void* topDetections)
{
    gtdLaunchConfig lc = gtdLaunchConfig(DT_BBOX, DT_SCORE);
    for (unsigned i = 0; i < gtdFuncVec.size(); ++i)
    {
        if (lc == gtdFuncVec[i])
        {
            DEBUG_PRINTF("gatherTopDetectionsOpt kernel %d\n", i);
            return gtdFuncVec[i].function(stream, shareLocation, clip, numImages, numPredsPerClass, numClasses, topK,
                keepTopK, indices, scores, bboxData,
                // keepCount,
                topDetections);
        }
    }
    return STATUS_BAD_PARAM;
}

} // namespace plugin
} // namespace nvinfer1
