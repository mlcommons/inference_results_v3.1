#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "ssdOpt.h"
#include "ssdOptMacros.h"
#include <hipcub/hipcub.hpp>
#include <hipDNN.h>

// debug
#include "nms_common.h"
#include <vector>

#define CUDA_MEM_ALIGN 256

// ALIGNPTR {{{
int8_t* alignPtr(int8_t* ptr, uintptr_t to)
{
    uintptr_t addr = (uintptr_t) ptr;
    if (addr % to)
    {
        addr += to - addr % to;
    }
    return (int8_t*) addr;
}
// }}}

// NEXTWORKSPACEPTR {{{
int8_t* nextWorkspacePtr(int8_t* ptr, uintptr_t previousWorkspaceSize)
{
    uintptr_t addr = (uintptr_t) ptr;
    addr += previousWorkspaceSize;
    return alignPtr((int8_t*) addr, CUDA_MEM_ALIGN);
}
// }}}

// CALCULATE TOTAL WORKSPACE SIZE {{{
size_t calculateTotalWorkspaceSize(size_t* workspaces, int count)
{
    size_t total = 0;
    for (int i = 0; i < count; i++)
    {
        total += workspaces[i];
        if (workspaces[i] % CUDA_MEM_ALIGN)
        {
            total += CUDA_MEM_ALIGN - (workspaces[i] % CUDA_MEM_ALIGN);
        }
    }
    return total;
}
// }}}

namespace nvinfer1
{
namespace plugin
{

ssdStatus_t detectionInferenceOpt(hipStream_t stream, const int N, const int C1, const int C2,
    const bool shareLocation, const bool varianceEncodedInTarget, const int backgroundLabelId,
    const int numPredsPerClass, const int numClasses, const int topK, const int keepTopK,
    const float confidenceThreshold, const float nmsThreshold, const CodeTypeSSD codeType, const DType_t DT_BBOX,
    const void* const* locData, const void* priorData, const DType_t DT_SCORE, const void* const* confData,
    void* topDetections, void* workspace, bool isNormalized, bool confSigmoid, bool confSoftmax,
    bool permuteBeforeReshape, bool concatInputs, const int numLayers, const int* featureSize, const int* numAnchors,
    const int* boxChannels, const int* confChannels, const bool packed32NCHW, hipdnnHandle_t cudnnHandle,
    hipdnnTensorDescriptor_t inScoreDesc, hipdnnTensorDescriptor_t outScoreDesc)
{
    // if we want to clip bbox output to [0,1]
    bool clip = true;

    const int locCount = N * C1;
    const bool clipBBox = false;
    const int numLocClasses = shareLocation ? 1 : numClasses;

    size_t bboxDataSize = detectionForwardBBoxDataSize(N, C1, DataType::kFLOAT);
    void* bboxDataRaw = workspace;

    ssdStatus_t status;

    // *******************************************************************
    // implmenting fp16

    // hardcode, since only need for development
    // NETWORK_SSD_MOBILE, TF_CENTER
    std::vector<int> dataCounts = {106463232, 58982400, 14745600, 5308416, 2359296, 589824};
    // NETWORK_SSD_RESNET34, CENTER_SIZE, softmax is on
    dataCounts = {56320000, 20480000, 5537792, 1605632, 202752, 202752};

    // locData, priorData
    bool isFp16Loc = false;
    bool isFp16Conf = false;

    auto dataTypeLoc = (isFp16Loc) ? DataType::kHALF : DataType::kFLOAT;
    __half** locDataFp16 = new __half*[numLayers];
    __half* priorDataFp16;
    __half* bboxDataRawFp16;
    if (isFp16Loc)
    {
        hipDeviceSynchronize();
        for (int i = 0; i < numLayers; i++)
        {
            CUDA_CHECK(hipMalloc(&locDataFp16[i], dataCounts[i] * sizeof(__half)));
            fp32_to_fp16(locDataFp16[i], reinterpret_cast<const float*>(locData[i]), dataCounts[i], stream);

            // fp16_to_fp32(reinterpret_cast<float*>(locData[i]), locDataFp16, localDataCounts[i], stream);

            DEBUG_PRINTF("decode: layer[%d] dataCounts= %d\n", i, dataCounts[i]);
        }

        DEBUG_PRINTF("Box encoding type: %d\n", codeType);

        size_t priorCount = (varianceEncodedInTarget) ? 4 * numPredsPerClass : 8 * numPredsPerClass;
        CUDA_CHECK(hipMalloc(&priorDataFp16, ((priorCount + 256 - 1) / 256) * 256 * sizeof(__half)));
        fp32_to_fp16(priorDataFp16, reinterpret_cast<const float*>(priorData), priorCount, stream);

        CUDA_CHECK(hipMalloc(&bboxDataRawFp16, 4 * locCount * sizeof(__half)));
    }

    const void* const* locDataT = (isFp16Loc) ? reinterpret_cast<const void* const*>(locDataFp16) : locData;
    const void* priorDataT = (isFp16Loc) ? reinterpret_cast<const void*>(priorDataFp16) : priorData;
    void* bboxDataRawT = (isFp16Loc) ? reinterpret_cast<void*>(bboxDataRawFp16) : bboxDataRaw;

    DEBUG_PRINTF("numthreads = %d\n", locCount);

    // *******************************************************************

    status = decodeBBoxesOpt(stream, locCount, codeType, varianceEncodedInTarget, numPredsPerClass, shareLocation,
        numLocClasses, backgroundLabelId, clipBBox, dataTypeLoc, locDataT, priorDataT, bboxDataRawT, numLayers,
        featureSize, numAnchors, boxChannels, confChannels, packed32NCHW,
        !permuteBeforeReshape /*softmax means reshape_before_permute*/, concatInputs);

    SSD_ASSERT_FAILURE(status == STATUS_SUCCESS);

    // fp32_to_fp16(bboxDataRawFp16,
    //             reinterpret_cast<const float*> (bboxDataRaw), 4 * locCount, stream);
    // fp16_to_fp32(reinterpret_cast<float*>(bboxDataRaw), bboxDataRawFp16, 4 * locCount, stream);

    if (isFp16Loc)
    {
        fp16_to_fp32(reinterpret_cast<float*>(bboxDataRaw), bboxDataRawFp16, 4 * locCount, stream);
    }

    // float for now
    void* bboxData;
    size_t bboxPermuteSize = detectionForwardBBoxPermuteSize(shareLocation, N, C1, DataType::kFLOAT);
    void* bboxPermute = nextWorkspacePtr((int8_t*) bboxDataRaw, bboxDataSize);

    SSD_ASSERT_FAILURE(shareLocation);
    bboxData = bboxDataRaw;

    size_t temp_active_counts_size, temp_sort_scores_size, temp_sort_indicies_size, temp_cub_storage_bytes;
    temp_cub_storage_bytes = topKScoresPerClassWorkspaceSize(N, numClasses, numPredsPerClass, topK, DT_SCORE,
        temp_active_counts_size, temp_sort_scores_size, temp_sort_indicies_size);
    temp_cub_storage_bytes
        = temp_cub_storage_bytes - temp_active_counts_size - temp_sort_scores_size - temp_sort_indicies_size;

    const int numScores = N * C2;
    size_t scoresSize = std::max(detectionForwardPreNMSSize(N, C2), temp_sort_scores_size);
    void* scores = nextWorkspacePtr((int8_t*) bboxPermute, bboxPermuteSize);
    void* softmaxScores = nextWorkspacePtr((int8_t*) scores, scoresSize);
    void* temp_scores = nullptr;

    size_t indicesSize = std::max(detectionForwardPreNMSSize(N, C2), temp_sort_indicies_size);
    void* indices = nextWorkspacePtr((int8_t*) softmaxScores, scoresSize);

    void* temp_indices = nextWorkspacePtr((int8_t*) indices, indicesSize);
    void* temp_active_counts = nextWorkspacePtr((int8_t*) temp_indices, indicesSize);
    void* temp_cub_storage = nextWorkspacePtr((int8_t*) temp_active_counts, temp_active_counts_size);

    size_t postNMSScoresSize = detectionForwardPostNMSSize(N, numClasses, topK);
    size_t postNMSIndicesSize = detectionForwardPostNMSSize(N, numClasses, topK);
    void* postNMSScores = nextWorkspacePtr((int8_t*) temp_cub_storage, temp_cub_storage_bytes);
    void* postNMSIndices = nextWorkspacePtr((int8_t*) postNMSScores, postNMSScoresSize);

    size_t numSegments = N * numClasses;
    size_t activeCountSize = numSegments * sizeof(int);
    void* activeCount = nextWorkspacePtr((int8_t*) postNMSIndices, postNMSIndicesSize);

    // to reduce work we want to know the amount of active elements per class after allClassNMS
    size_t activeCountPerBatchSize = N * sizeof(int);
    void* activeCountPerBatch = nextWorkspacePtr((int8_t*) activeCount, activeCountSize);

    void* sortingWorkspace = nextWorkspacePtr((int8_t*) activeCountPerBatch, activeCountPerBatchSize);

    // *******************************************************************
    // set up conf buffers
    DEBUG_PRINTF("numScores = %d\n", numScores);

    auto dataTypeConf = (isFp16Conf) ? DataType::kHALF : DataType::kFLOAT;
    __half** confDataFp16 = new __half*[numLayers];
    __half* scoresFp16;
    if (isFp16Loc)
    {
        hipDeviceSynchronize();
        for (int i = 0; i < numLayers; i++)
        {
            CUDA_CHECK(hipMalloc(&confDataFp16[i], dataCounts[i] * sizeof(__half)));
            fp32_to_fp16(confDataFp16[i], reinterpret_cast<const float*>(confData[i]), dataCounts[i], stream);
        }

        CUDA_CHECK(hipMalloc(&scoresFp16, numScores * sizeof(__half)));
    }

    const void* const* confDataT = (isFp16Conf) ? reinterpret_cast<const void* const*>(confDataFp16) : confData;

    void* scoresT = (isFp16Conf) ? reinterpret_cast<void*>(scoresFp16) : scores;
    // *******************************************************************

    // that is what we currently support
    // assert(confSoftmax && !permuteBeforeReshape || !confSoftmax);
    // need a conf_scores
    // TODO Add support for both permutations
    if (confSoftmax && SSD_RETINA_NET == 0)
    { // confSoftmax
        DEBUG_PRINTF("Forward: permuteBeforeReshape = %d\n", permuteBeforeReshape);

        status = permuteConfData(stream, numScores, numClasses, numPredsPerClass, 1, dataTypeConf, confSigmoid,
            confDataT, scoresT, activeCount, numLayers, featureSize, numAnchors, boxChannels, permuteBeforeReshape,
            concatInputs, packed32NCHW);
        SSD_ASSERT_FAILURE(status == STATUS_SUCCESS);

        if (isFp16Conf)
        {
            fp16_to_fp32(reinterpret_cast<float*>(scores), scoresFp16, numScores, stream);
        }
    }

    if (confSoftmax)
    {
        status = softmaxScore(stream, N, numClasses, numPredsPerClass, 1, DataType::kFLOAT, scores, softmaxScores,
            cudnnHandle, inScoreDesc, outScoreDesc);
        SSD_ASSERT_FAILURE(status == STATUS_SUCCESS);

        temp_scores = scores;
        scores = softmaxScores;
    }
    else
    {
        temp_scores = softmaxScores;
    }

    // if(!permuteBeforeReshape || confSoftmax)
    if (SSD_RETINA_NET == 1 || confSoftmax)
    {
        status = topKScoresPerClass(stream, N, numClasses, numPredsPerClass, topK, backgroundLabelId,
            confidenceThreshold, DataType::kFLOAT, scores, indices, activeCount, activeCountPerBatch, temp_scores,
            temp_indices, temp_active_counts, temp_cub_storage_bytes, temp_cub_storage, numPredsPerClass, 1,
            confSigmoid, confData, numLayers, featureSize, numAnchors, boxChannels, packed32NCHW);
        // sortingWorkspace);
    }
    else
    {
        assert(concatInputs == true);
        status = topKScoresPerClassFusedPermute(stream, N, numClasses, numPredsPerClass, topK, backgroundLabelId,
            confidenceThreshold, DataType::kFLOAT, scores, indices, activeCount, activeCountPerBatch, sortingWorkspace,
            numPredsPerClass, 1, confSigmoid, confData, numLayers, featureSize, numAnchors, boxChannels, packed32NCHW);
    }

    SSD_ASSERT_FAILURE(status == STATUS_SUCCESS);

    status = allClassNMSOpt(stream, N, numClasses, numPredsPerClass, topK, nmsThreshold, shareLocation, isNormalized,
        DataType::kFLOAT, DataType::kFLOAT, bboxData, scores, indices, postNMSScores, postNMSIndices, activeCount,
        activeCountPerBatch, false);
    SSD_ASSERT_FAILURE(status == STATUS_SUCCESS);

    status = topKScoresPerImage(stream, N, numClasses * topK, topK, DataType::kFLOAT, postNMSScores, postNMSIndices,
        scores, indices, activeCount, activeCountPerBatch, temp_active_counts, temp_cub_storage_bytes,
        temp_cub_storage);
    // sortingWorkspace);
    SSD_ASSERT_FAILURE(status == STATUS_SUCCESS);

    status = gatherTopDetectionsOpt(stream, shareLocation, clip, N, numPredsPerClass, numClasses, topK, keepTopK,
        DataType::kFLOAT, DataType::kFLOAT, indices, scores, bboxData, topDetections);
    SSD_ASSERT_FAILURE(status == STATUS_SUCCESS);
    return STATUS_SUCCESS;

    // *******************************************************************
    // implmenting fp16
    if (isFp16Loc)
    {
        hipDeviceSynchronize();
        for (int i = 0; i < numLayers; i++)
        {
            hipFree(locDataFp16[i]);
        }
        hipFree(priorDataFp16);
        hipFree(bboxDataRawFp16);
    }

    if (isFp16Conf)
    {
        hipDeviceSynchronize();
        for (int i = 0; i < numLayers; i++)
        {
            hipFree(confDataFp16[i]);
        }
        hipFree(scoresFp16);
    }

    // *******************************************************************
}

} // namespace plugin
} // namespace nvinfer1
