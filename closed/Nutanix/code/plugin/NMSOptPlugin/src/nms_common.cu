#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "nms_common.h"

__global__ void fp32_to_fp16_kernel(__half* dst, const float* src, int count)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= count)
        return;

    dst[idx] = __float2half_rn(src[idx]);
}

__global__ void fp16_to_fp32_kernel(float* dst, const __half* src, int count)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= count)
        return;

    dst[idx] = __half2float(src[idx]);
}

void fp32_to_fp16(__half* dst, const float* src, int count, hipStream_t stream)
{

    const int BLOCK_SIZE = 256;
    int grid_dim = (count + BLOCK_SIZE - 1) / BLOCK_SIZE;

    fp32_to_fp16_kernel<<<grid_dim, BLOCK_SIZE, 0, stream>>>(dst, src, count);
}

void fp16_to_fp32(float* dst, const __half* src, int count, hipStream_t stream)
{

    const int BLOCK_SIZE = 256;
    int grid_dim = (count + BLOCK_SIZE - 1) / BLOCK_SIZE;

    fp16_to_fp32_kernel<<<grid_dim, BLOCK_SIZE, 0, stream>>>(dst, src, count);
}
