#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hipcub/hipcub.hpp>
#include <vector>

#include <fstream>
#include <iostream>

#include "nms_common.h"
#include "ssdOpt.h"
#include "ssdOptMacros.h"
#include "topK.h"

// C-API for the async copy
#include "nms_common.h"
#include <cuda_pipeline.h>

#define CUSTOM_TRANSPOSE 1
#define ENABLE_FUSED_TRANSPOSE 1

#if CUSTOM_TRANSPOSE == 0
#include <hipblas.h>
#endif

template <typename T>
void saveDeviceBuffer_topk(const T* buf, size_t count, std::ofstream& output)
{
    T* hBuf = new T[count];
    hipDeviceSynchronize();
    hipMemcpy(hBuf, buf, count * sizeof(T), hipMemcpyDeviceToHost);
    output.write((char*) hBuf, count * sizeof(T));
    delete[] hBuf;
}

namespace nvinfer1
{
namespace plugin
{

template <int TILE_X, int TILE_Y, int BLOCK_DIM_X, int BLOCK_DIM_Y>
__global__ void batched_transpose_fuse_sigmoid_topk_prepare_kernel(const float* data_in, float* data_out,
    const int num_classes, const int num_priors, int* active_counts, int* out_indices, int background_class_id,
    float threshold)
{
    // num_classes is "fast" dimension in data_in
    // BLOCK_DIM_X == TILE_X

    constexpr int SMEM_STRIDE = (TILE_X < TILE_Y) ? TILE_Y + 1 : TILE_X + 1;
    __shared__ float smem[SMEM_STRIDE * TILE_Y];
    __shared__ int smem_idx[SMEM_STRIDE * TILE_Y];

    __shared__ int smem_offset[BLOCK_DIM_X * (BLOCK_DIM_Y + 1)];
    __shared__ int smem_global_offset[TILE_X];

    // batch id
    const int n = blockIdx.z;
    const int n_offset = n * num_classes * num_priors;

    const int tile_x_in_offset = blockIdx.x * TILE_X;
    const int tile_y_in_offset = blockIdx.y * TILE_Y;

    const float* data_in_tile = data_in + n_offset + tile_x_in_offset + tile_y_in_offset * num_classes;

    if (tile_x_in_offset >= num_classes || tile_y_in_offset >= num_priors)
    {
        return;
    }

    int class_id = tile_x_in_offset + threadIdx.x;
    float is_not_background_class_id = (class_id != background_class_id) ? 1.0F : 0.0F;

    int active_count_per_y = 0;

    const int SCORES_PER_THREAD = TILE_Y / BLOCK_DIM_Y;
    float reg_scores[SCORES_PER_THREAD] = {0};

    //__syncthreads();

    // first element of the prefix sum
    smem_offset[threadIdx.y * BLOCK_DIM_X + threadIdx.x] = 0;

#pragma unroll
    for (int jj = 0; jj < SCORES_PER_THREAD; jj++)
    {
        int j = jj * BLOCK_DIM_Y;
        int x_idx = threadIdx.x;
        int y_idx = j + threadIdx.y;
        bool is_not_valid = tile_x_in_offset + x_idx >= num_classes || tile_y_in_offset + y_idx >= num_priors;
        // int tile_in_idx = threadIdx.x + tile_x_in_offset
        float score = (is_not_valid) ? -30.F : data_in_tile[y_idx * num_classes + x_idx];
        score = __expf(score) / (1.0f + __expf(score));
        // score = (score < threshold)? 0.0F : is_not_background_class_id * score;
        is_not_valid = is_not_valid || (class_id == background_class_id) || score < threshold;
        bool is_valid = !is_not_valid;
        active_count_per_y += is_valid;
        reg_scores[jj] = (is_valid) ? score : 0.0F;

        // smem[SMEM_STRIDE * y_idx + x_idx] = (is_not_valid)? 0.F : data_in_tile[y_idx * num_classes + x_idx];
        // if (is_valid) {
        //     smem[SMEM_STRIDE * (j + active_count_per_y) + x_idx] = score;
        // }
    }

    // prefix sum
    smem_offset[(threadIdx.y + 1) * BLOCK_DIM_X + threadIdx.x] = active_count_per_y;

    __syncthreads();

    if (threadIdx.y == 0)
    {
        for (int j = 2; j <= BLOCK_DIM_Y; j++)
        {
            smem_offset[j * BLOCK_DIM_X + threadIdx.x] += smem_offset[(j - 1) * BLOCK_DIM_X + threadIdx.x];
        }
    }

    __syncthreads();

    int partial_offset = smem_offset[threadIdx.y * BLOCK_DIM_X + threadIdx.x];
    int offset = smem_offset[BLOCK_DIM_Y * BLOCK_DIM_X + threadIdx.x];
    if (class_id < num_classes)
    {
        if (threadIdx.y == 0)
        {
            smem_global_offset[threadIdx.x] = atomicAdd(&active_counts[n * num_classes + class_id], offset);
        }
    }
    int cur = 0;
#pragma unroll
    for (int jj = 0; jj < SCORES_PER_THREAD; jj++)
    {
        int j = jj * BLOCK_DIM_Y;
        int offset = smem_offset[threadIdx.y * BLOCK_DIM_X + threadIdx.x];
        if (reg_scores[jj] != 0.0F)
        {
            smem[SMEM_STRIDE * threadIdx.x + partial_offset + cur] = reg_scores[jj];
            smem_idx[SMEM_STRIDE * threadIdx.x + partial_offset + cur]
                = class_id * num_priors + tile_y_in_offset + j + threadIdx.y;
            cur++;
        }
    }

    __syncthreads();

    int tile_x_out_offset = 0;
    int tile_y_out_offset = blockIdx.x * TILE_X;

#pragma unroll
    for (int j = 0; j < TILE_X; j += BLOCK_DIM_Y)
    {
        int y_idx = j + threadIdx.y;
        tile_x_out_offset = smem_global_offset[y_idx];
        float* data_out_tile = data_out + n_offset + tile_x_out_offset + tile_y_out_offset * num_priors;
        int* out_indices_tile = out_indices + n_offset + tile_x_out_offset + tile_y_out_offset * num_priors;
        // TILE_X == BLOCK_DIM_X
        offset = smem_offset[BLOCK_DIM_Y * BLOCK_DIM_X + y_idx];
        for (int x_idx = threadIdx.x; x_idx < offset; x_idx += BLOCK_DIM_X)
        {
            bool is_not_valid = tile_x_out_offset + x_idx >= num_priors // remove
                || tile_y_out_offset + y_idx >= num_classes;
            if (!is_not_valid)
            {
                // if (threadIdx.x == 0 && tile_x_out_offset != 0) {
                // if (smem[SMEM_STRIDE * x_idx + y_idx] > 0.89f) {
                //     printf("%f %d %d %d\n", smem[SMEM_STRIDE * x_idx + y_idx], smem_idx[SMEM_STRIDE * x_idx + y_idx],
                //     x_idx, y_idx);
                // }
                data_out_tile[y_idx * num_priors + x_idx] = smem[SMEM_STRIDE * y_idx + x_idx];
                out_indices_tile[y_idx * num_priors + x_idx]
                    = smem_idx[SMEM_STRIDE * y_idx + x_idx]; // tile_x_out_offset + x_idx;
            }
        }
    }
}

template <int BLOCK_THREADS>
__global__ void get_cub_offsets_kernel(
    int* in_active_counts, int* out_begin_offsets, int* out_end_offsets, int items, int segments, int num_top_k)
{
    // in_active_counts and out_end_offsets can be the same buffer

    int segment_id = blockIdx.x * BLOCK_THREADS + threadIdx.x;
    const int stride = items / segments;
    if (segment_id >= segments)
        return;

    out_end_offsets[segment_id] = in_active_counts[segment_id] + segment_id * stride;

    out_begin_offsets[segment_id] = segment_id * stride;
}

template <int TILE_X, int TILE_Y, int BLOCK_DIM_X, int BLOCK_DIM_Y>
__global__ void batched_transpose_kernel(
    const float* data_in, float* data_out, const int cols, const int rows, const int output_n_stride)
{
    // cols is "fast" dimension in data_in (i.e. row major)
    // BLOCK_DIM_X == TILE_X

    constexpr int SMEM_STRIDE = TILE_X + 1;
    __shared__ float smem[SMEM_STRIDE * TILE_Y];

    // batch id
    const int n = blockIdx.z;
    const int n_in_offset = n * cols * rows; // n * cols * rows;
    const int n_out_offset = n * output_n_stride;

    const int tile_x_in_offset = blockIdx.x * TILE_X;
    const int tile_y_in_offset = blockIdx.y * TILE_Y;

    const float* data_in_tile = data_in + n_in_offset + tile_x_in_offset + tile_y_in_offset * cols;

    if (tile_x_in_offset >= cols || tile_y_in_offset >= rows)
    {
        return;
    }

#pragma unroll
    for (int j = 0; j < TILE_Y; j += BLOCK_DIM_Y)
    {
        int x_idx = threadIdx.x;
        int y_idx = j + threadIdx.y;
        bool is_not_valid = tile_x_in_offset + x_idx >= cols || tile_y_in_offset + y_idx >= rows;
        // int tile_in_idx = threadIdx.x + tile_x_in_offset
        smem[SMEM_STRIDE * y_idx + x_idx] = (is_not_valid) ? 0.F : data_in_tile[y_idx * cols + x_idx];
    }

    __syncthreads();

    int tile_x_out_offset = blockIdx.y * TILE_Y;
    int tile_y_out_offset = blockIdx.x * TILE_X;
    float* data_out_tile = data_out + n_out_offset + tile_x_out_offset + tile_y_out_offset * rows;

#pragma unroll
    for (int x_idx = threadIdx.x; x_idx < TILE_Y; x_idx += BLOCK_DIM_X)
    {
#pragma unroll
        for (int j = 0; j < TILE_X; j += BLOCK_DIM_Y)
        {
            int y_idx = j + threadIdx.y;
            bool is_not_valid = tile_x_out_offset + x_idx >= rows || tile_y_out_offset + y_idx >= cols;
            if (!is_not_valid)
            {
                data_out_tile[y_idx * rows + x_idx] = smem[SMEM_STRIDE * x_idx + y_idx];
            }
        }
    }
}

#define ENABLE_LDGSTS 1
template <int TILE_X, int TILE_Y, int BLOCK_DIM_X, int BLOCK_DIM_Y, int N_TILES = 1>
__global__ void batched_transpose_pf_kernel(
    const float* data_in, float* data_out, const int cols, const int rows, const int output_n_stride)
{
    // cols is "fast" dimension in data_in
    // BLOCK_DIM_X == TILE_X
    // N_TILES = number of tiles in Y direction (we have a very narrow matrix)

    constexpr int SMEM_STRIDE = TILE_X + 1;
    constexpr int SMEM_FACTOR = (N_TILES > 1) ? 2 : 1;
    __shared__ float smem[SMEM_FACTOR * SMEM_STRIDE * TILE_Y];

    // batch id
    const int n = blockIdx.z;
    const int n_in_offset = n * cols * rows; // n * cols * rows;
    const int n_out_offset = n * output_n_stride;

    const int tile_x_in_offset = blockIdx.x * TILE_X;
    const int tile_y_in_offset = blockIdx.y * TILE_Y * N_TILES;

    if (tile_x_in_offset >= cols || tile_y_in_offset >= rows)
    {
        return;
    }

    const float* data_in_tile = data_in + n_in_offset + tile_x_in_offset + tile_y_in_offset * cols;
#pragma unroll
    for (int j = 0; j < TILE_Y; j += BLOCK_DIM_Y)
    {
        int x_idx = threadIdx.x;
        int y_idx = j + threadIdx.y;
        bool is_not_valid = tile_x_in_offset + x_idx >= cols || tile_y_in_offset + y_idx >= rows;
        // int tile_in_idx = threadIdx.x + tile_x_in_offset
#if ENABLE_LDGSTS == 1
        size_t zfill = (is_not_valid) ? sizeof(float) : 0;
        __pipeline_memcpy_async(
            &smem[SMEM_STRIDE * y_idx + x_idx], &data_in_tile[y_idx * cols + x_idx], sizeof(float), zfill);
#else
        smem[SMEM_STRIDE * y_idx + x_idx] = (is_not_valid) ? 0.F : data_in_tile[y_idx * cols + x_idx];
#endif
    }

#if ENABLE_LDGSTS == 1
    __pipeline_commit();
#endif

    int istage = 0;
    int istage_next = 1;
    for (int itile = 0; itile < N_TILES; itile++)
    {

#if ENABLE_LDGSTS == 1
        __pipeline_wait_prior(0);
#endif
        __syncthreads();

        istage_next = (istage + 1) % 2;

        const int tile_y_in_offset = blockIdx.y * TILE_Y * N_TILES + (itile + 1) * TILE_Y;
        const float* data_in_tile = data_in + n_in_offset + tile_x_in_offset + tile_y_in_offset * cols;
        if (tile_y_in_offset < rows && itile < N_TILES - 1)
        {
#pragma unroll
            for (int j = 0; j < TILE_Y; j += BLOCK_DIM_Y)
            {
                int x_idx = threadIdx.x;
                int y_idx = j + threadIdx.y;
                bool is_not_valid = tile_x_in_offset + x_idx >= cols || tile_y_in_offset + y_idx >= rows;
#if ENABLE_LDGSTS == 1
                size_t zfill = (is_not_valid) ? sizeof(float) : 0;
                __pipeline_memcpy_async(&smem[istage_next * SMEM_STRIDE * TILE_Y + SMEM_STRIDE * y_idx + x_idx],
                    &data_in_tile[y_idx * cols + x_idx], sizeof(float), zfill);
#else
                smem[istage_next * SMEM_STRIDE * TILE_Y + SMEM_STRIDE * y_idx + x_idx]
                    = (is_not_valid) ? 0.F : data_in_tile[y_idx * cols + x_idx];
#endif
            }
#if ENABLE_LDGSTS == 1
            __pipeline_commit();
#endif
        }

        // transpose the previous stage while loading the next one
        int tile_x_out_offset = blockIdx.y * TILE_Y * N_TILES + itile * TILE_Y;
        int tile_y_out_offset = blockIdx.x * TILE_X;
        float* data_out_tile = data_out + n_out_offset + tile_x_out_offset + tile_y_out_offset * rows;

#pragma unroll
        for (int x_idx = threadIdx.x; x_idx < TILE_Y; x_idx += BLOCK_DIM_X)
        {
#pragma unroll
            for (int j = 0; j < TILE_X; j += BLOCK_DIM_Y)
            {
                int y_idx = j + threadIdx.y;
                bool is_not_valid = tile_x_out_offset + x_idx >= rows || tile_y_out_offset + y_idx >= cols;
                if (!is_not_valid)
                {
                    data_out_tile[y_idx * rows + x_idx]
                        = smem[istage * SMEM_STRIDE * TILE_Y + SMEM_STRIDE * x_idx + y_idx];
                }
            }
        }

        istage = istage_next;
    }
}

// __global__ void sigmoid_kernel(float* data, int nthreads) {
//     const int tid = blockIdx.x * blockDim.x + threadIdx.x;

//     if (tid >= nthreads) return;

//     float val = data[tid];
//     data[tid] = __expf(val) / (1.0f + __expf(val));
// }

#if CUSTOM_TRANSPOSE == 0
hipblasHandle_t handle_;
#endif

void permuteConfDataFuseCubTopKPrepare(hipStream_t stream, const int nthreads, const int num_classes,
    const int num_priors, int num_layers, bool confSigmoid, void* new_data, void* tmp_data,
    const void* const* conf_data, void* active_counts_per_class, const int* feature_size, const int* num_anchors,
    const bool packed32_nchw, void* out_indices, const int background_class_id, const float threshold)
{

    assert(packed32_nchw == 0);

#if CUSTOM_TRANSPOSE == 0
    static bool is_first_call = true;
    if (is_first_call)
    {
        hipblasCreate(&handle_);
    }
    hipblasSetStream(handle_, stream);
#endif

    float alpha = 1.f;
    float beta = 0.f;
    float* b_matrix = 0;

    int output_n_stride = num_classes * num_priors;
    int num_images = nthreads / output_n_stride;
    // printf("permuteConfData_transpose: num_images = %d, num_classes = %d, output_n_stride = %d, \n", num_images,
    // num_classes, output_n_stride);

    hipMemsetAsync(active_counts_per_class, 0, sizeof(int), stream);

    // printf("enter cublas\n");

    constexpr int BLOCK_DIM_X = 32;
    constexpr int BLOCK_DIM_Y = 8;
    constexpr int TILE_X = BLOCK_DIM_X;
    constexpr int TILE_Y = 32;
    constexpr int N_TILES = 1;

#if CUSTOM_TRANSPOSE == 1

    int output_concat_offset = 0;
    for (int layer = 0; layer < num_layers; layer++)
    {
        int layer_hw_size = feature_size[layer] * feature_size[layer];
        int prev_layer_prior_size
            = (layer == 0) ? 0 : num_anchors[layer - 1] * feature_size[layer - 1] * feature_size[layer - 1];
        output_concat_offset += prev_layer_prior_size * num_classes;

        int m = num_anchors[layer] * num_classes; // number of conf channels
        int n = layer_hw_size;

        dim3 block = dim3(BLOCK_DIM_X, BLOCK_DIM_Y, 1);
        dim3 grid = dim3(div_up(n, TILE_X), div_up(m, TILE_Y * N_TILES), num_images);
        batched_transpose_pf_kernel<TILE_X, TILE_Y, BLOCK_DIM_X, BLOCK_DIM_Y, N_TILES>
            <<<grid, block, 0, stream>>>(reinterpret_cast<const float*>(conf_data[layer]),
                reinterpret_cast<float*>(tmp_data) + output_concat_offset, n, m, output_n_stride);

        if (layer == num_layers - 1)
        {
            assert(output_concat_offset + num_anchors[layer] * layer_hw_size * num_classes == num_priors * num_classes);
        }
    }
#endif

#if CUSTOM_TRANSPOSE == 0
    for (int ib = 0; ib < num_images; ib++)
    {
        int output_n_offset = ib * output_n_stride;
        int output_concat_offset = output_n_offset;
        for (int layer = 0; layer < num_layers; layer++)
        {

            int layer_hw_size = feature_size[layer] * feature_size[layer];
            int prev_layer_prior_size
                = (layer == 0) ? 0 : num_anchors[layer - 1] * feature_size[layer - 1] * feature_size[layer - 1];
            output_concat_offset += prev_layer_prior_size * num_classes;
            // printf("permuteConfData_cublas: layer_hw_size = %d, num_anchors = %d, prev_layer_prior_size = %d,
            // output_concat_offset = %d\n",layer_hw_size,  num_anchors[layer], prev_layer_prior_size,
            // output_concat_offset);
            // cublas implies column major
            // number of rows of transposed input(in column major), i.e. output
            int m = num_anchors[layer] * num_classes; // number of conf channels
            // number of columns(in column major)
            int n = layer_hw_size;
            int lda = layer_hw_size;
            int ldb = m;
            int ldc = m;
            hipblasSgeam(handle_, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, &alpha,
                reinterpret_cast<const float*>(conf_data[layer]) + ib * m * n, lda, &beta, b_matrix, ldb,
                reinterpret_cast<float*>(tmp_data) + output_concat_offset, ldc);

            if (layer == num_layers - 1)
            {
                assert(output_concat_offset + num_anchors[layer] * layer_hw_size * num_classes - output_n_offset
                    == num_priors * num_classes);
            }
        }

#if (ENABLE_FUSED_TRANSPOSE == 0)
        // transpose
        int m = num_priors;
        int n = num_classes;
        int lda = n;
        int ldb = m;
        int ldc = m;
        hipblasSgeam(handle_, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, &alpha,
            reinterpret_cast<float*>(tmp_data) + output_n_offset, lda, &beta, b_matrix, ldb,
            reinterpret_cast<float*>(new_data) + output_n_offset, ldc);
#endif
    }
#endif

#if (ENABLE_FUSED_TRANSPOSE == 1)

#if 0
    dim3 block = dim3(BLOCK_DIM_X, BLOCK_DIM_Y, 1);
    dim3 grid = dim3(div_up(num_classes, TILE_X), div_up(num_priors,TILE_Y), num_images);
    batched_transpose_kernel<TILE_X, TILE_Y, BLOCK_DIM_X, BLOCK_DIM_Y><<<grid, block, 0, stream>>>
            (reinterpret_cast<float*>(tmp_data),
             reinterpret_cast<float*>(new_data),
             num_classes,
             num_priors);
#else
#if ENABLE_FUSED_TRANSPOSE == 1
    dim3 block = dim3(BLOCK_DIM_X, BLOCK_DIM_Y, 1);
    dim3 grid = dim3(div_up(num_classes, TILE_X), div_up(num_priors, TILE_Y), num_images);
    hipMemsetAsync(active_counts_per_class, 0, num_images * num_classes * sizeof(int), stream);
    batched_transpose_fuse_sigmoid_topk_prepare_kernel<TILE_X, TILE_Y, BLOCK_DIM_X, BLOCK_DIM_Y>
        <<<grid, block, 0, stream>>>(reinterpret_cast<const float*>(tmp_data), reinterpret_cast<float*>(new_data),
            num_classes, num_priors, reinterpret_cast<int*>(active_counts_per_class),
            reinterpret_cast<int*>(out_indices), background_class_id, threshold);
#endif

#endif
#endif

#if SSD_CUBLAS_CONF_PERMUTE == 0
    int block_size_sigmoid = 256;
    int grid = div_up(nthreads, block_size_sigmoid);
    sigmoid_kernel<<<div_up(nthreads, block_size_sigmoid), block_size_sigmoid, 0, stream>>>(
        reinterpret_cast<float*>(new_data), nthreads);
#endif
}

//#ifdef SSD_STABLE_TOPK
struct BlockPrefixCallbackOp
{
    // Running prefix
    int running_total;
    // Constructor
    __device__ BlockPrefixCallbackOp(int running_total)
        : running_total(running_total)
    {
    }
    // Callback operator to be entered by the first warp of threads in the block.
    // Thread-0 is responsible for returning a value for seeding the block-wide scan.
    __device__ int operator()(int block_aggregate)
    {
        int old_prefix = running_total;
        running_total += block_aggregate;
        return old_prefix;
    }
};
//#endif

#if USE_CUB_SEGMENTED_SORT == 1

// TODO: implement reduction of to get real actrive counts
template <typename T_SCORE, int BLOCK_THREADS>
__global__ void cub_top_k_prepare(T_SCORE* in_scores, T_SCORE* out_scores, int* out_indices, int* begin_offsets,
    int* end_offsets, int* active_count_per_batch, int items, int segments, int background_class_id, float threshold)
{

    typedef hipcub::BlockScan<int, BLOCK_THREADS> BlockScan;
    __shared__ typename BlockScan::TempStorage temp_storage;
    // Initialize running total
    BlockPrefixCallbackOp prefix_op(0);

    if (threadIdx.x == 0)
    {
        // We have to initialize active_count_per_batch for the following allClassNMS kernel.
        // Do it here to avoid to avoid an extra memset launch.
        if (blockIdx.x == 0)
        {
            active_count_per_batch[blockIdx.y] = 0;
        }
    }

    const int class_id = blockIdx.x;
    const int segment = blockIdx.y * gridDim.x + blockIdx.x;
    const int stride = items / segments;
    const int begin_offset = segment * stride;

    in_scores += begin_offset;
    out_scores += begin_offset;
    out_indices += begin_offset;

    begin_offsets[segment] = segment * stride;
    // end_offsets[segment] = (segment + 1) * stride;

    if (class_id == background_class_id)
    {
        end_offsets[segment] = segment * stride;
        return;
    }

    float is_not_background_class_id = (class_id != background_class_id) ? 1.0F : 0.0F;
    int end = div_up(stride, BLOCK_THREADS) * BLOCK_THREADS;
    for (int idx = threadIdx.x; idx < end; idx += BLOCK_THREADS)
    {
#if SSD_CUBLAS_CONF_PERMUTE == 1
        T_SCORE score = (idx < stride) ? __expf(in_scores[idx]) / (1.0f + __expf(in_scores[idx])) : 0.0F;
#else
        T_SCORE score = (idx < stride) ? in_scores[idx] : 0.0F;
#endif
        score = (score < threshold) ? 0.0F : is_not_background_class_id * score;
        // debug
        // if (class_id == -1 && score > 0.0f) {
        //     printf("%d %f\n", idx, score);
        // }
        int selected = score > 0.0F;
        int offset;
        BlockScan(temp_storage).ExclusiveSum(selected, offset, prefix_op);

        if (selected)
        {
            out_scores[offset] = score;
            out_indices[offset] = idx + stride * blockIdx.x;
        }

        __syncthreads();
    }

    if (threadIdx.x == 0)
    {
        end_offsets[segment] = prefix_op.running_total + segment * stride;
    }

    // if (threadIdx.x == 0) {
    //     //printf("%d, %d, %d\n", segment, begin_offsets[segment], end_offsets[segment]);
    //     printf("%d, %d\n", segment, prefix_op.running_total);
    // }

    // if (threadIdx.x == 0)
    //     printf("%d, %d\n", segment, prefix_op.running_total);

    // assert(prefix_op.running_total <= stride);
}

template <int BLOCK_THREADS>
__global__ void get_active_counts(int* in_end_offsets, int* out_active_counts, int items, int segments, int num_top_k)
{

    int segment_id = blockIdx.x * BLOCK_THREADS + threadIdx.x;

    const int stride = items / segments;

    if (segment_id >= segments)
        return;

    out_active_counts[segment_id] = min(max(0, in_end_offsets[segment_id] - segment_id * stride), num_top_k);

    DEBUG_PRINTF("%d, %d\n", segment_id, min(max(0, in_end_offsets[segment_id] - segment_id * stride), num_top_k));
}

#endif

constexpr int TOPK_PER_CLASS_BLOCK_THREADS = 512;

namespace
{
// sort one segment per cta
template <typename T_SCORE, int BLOCK_THREADS, int ELEMENTS_PER_THREAD>
__global__ void blockSortKernel(const T_SCORE* d_keys_in, T_SCORE* d_keys_out, const int32_t* d_values_in,
    int32_t* d_values_out, const int32_t* active_counts, int num_items_, int stride_items, int num_segments)
{
    // Specialize BlockRadixSort for a 1D block
    typedef hipcub::BlockRadixSort<T_SCORE, BLOCK_THREADS, ELEMENTS_PER_THREAD, int32_t> BlockRadixSort;

    // Allocate shared memory for BlockRadixSort
    __shared__ typename BlockRadixSort::TempStorage temp_storage;

    if (blockIdx.x >= num_segments)
        return;

    // if (threadIdx.x == 0)
    //     DEBUG_PRINTF("active counts[%d] = %d\n", blockIdx.x, active_counts[blockIdx.x]);

    int num_items = active_counts[blockIdx.x] > num_items_ ? num_items_ : active_counts[blockIdx.x];

    if (num_items == 0)
    {
        return;
    }

    // Obtain a segment of consecutive items that are blocked across threads
    T_SCORE thread_keys[ELEMENTS_PER_THREAD];
    int32_t thread_values[ELEMENTS_PER_THREAD];

    int32_t block_offset = blockIdx.x * stride_items;
    hipcub::LoadDirectStriped<BLOCK_THREADS>(threadIdx.x, d_keys_out + block_offset, thread_keys, num_items, 0);
    hipcub::LoadDirectStriped<BLOCK_THREADS>(threadIdx.x, d_values_out + block_offset, thread_values, num_items, -1);
    __syncthreads();

    // Collectively sort the keys and values among block threads
    BlockRadixSort(temp_storage).SortDescendingBlockedToStriped(thread_keys, thread_values);

    // Store output in striped fashion
    hipcub::StoreDirectStriped<BLOCK_THREADS>(threadIdx.x, d_keys_out + block_offset, thread_keys, num_items);
    hipcub::StoreDirectStriped<BLOCK_THREADS>(threadIdx.x, d_values_out + block_offset, thread_values, num_items);
}

/// block sort kernel
template <typename T_SCORE>
void blockSort(const T_SCORE* d_keys_in, T_SCORE* d_keys_out, const int32_t* d_values_in, int32_t* d_values_out,
    const int32_t* active_counts, int num_items, int stride_items, int num_segments, hipStream_t stream)
{
    if (num_items == 0)
        return;

    int kernel_index = div_up(num_items, 128) - 1;
    int warps_per_cta = (kernel_index + 1) * 128 / 32;
    assert(warps_per_cta <= 32);

    dim3 block(warps_per_cta * 32);
    dim3 grid(num_segments);

    using kernel_func = void (*)(const T_SCORE* d_keys_in, T_SCORE* d_keys_out, const int32_t* d_values_in,
        int32_t* d_values_out, const int32_t* active_counts, int num_items, int stride_items, int num_segments);

    static const kernel_func kernel_funcs[] = {
        &blockSortKernel<T_SCORE, 128, 1>,
        &blockSortKernel<T_SCORE, 256, 1>,
        &blockSortKernel<T_SCORE, 384, 1>,
        &blockSortKernel<T_SCORE, 512, 1>,
        &blockSortKernel<T_SCORE, 640, 1>,
        &blockSortKernel<T_SCORE, 768, 1>,
        &blockSortKernel<T_SCORE, 896, 1>,
        &blockSortKernel<T_SCORE, 1024, 1>,
    };
    kernel_funcs[kernel_index]<<<grid, block, 0, stream>>>(
        d_keys_in, d_keys_out, d_values_in, d_values_out, active_counts, num_items, stride_items, num_segments);
}

template <int ITEMS_PER_THREAD, int BLOCK_THREADS = 512>
__global__ void top_k_cuda_fused_prepare(int* in, int* out, int* out_indices, int* active_counts_per_class,
    int* active_count_per_batch, int items, unsigned int num_top_k, int segments, int num_sub_segments,
    int background_class_id, float threshold)
{

    extern __shared__ int2 dynamic_smem[];
    int2* selected_elements = dynamic_smem;
#ifdef SSD_STABLE_TOPK
    int active_count;
    __shared__ unsigned int selected_count;
#else
    __shared__ unsigned int selected_count;
    // stores the number of elements which are above the threshold
    __shared__ unsigned int active_count;
#endif

#ifdef SSD_STABLE_TOPK
    // Specialize BlockScan type for our thread block
    typedef hipcub::BlockScan<int, BLOCK_THREADS> BlockScan;
    __shared__ typename BlockScan::TempStorage temp_storage;
    // Initialize running total
    BlockPrefixCallbackOp prefix_op(0);
#endif

    // int num_sub_segments = gridDim.z;
    // int sub_segment_id = blockIdx.z;

    unsigned int old_selected_count;

    // this is a workaround
    int class_id = blockIdx.x / num_sub_segments;
    int segment = blockIdx.y * gridDim.x + blockIdx.x;
    int stride = items * num_sub_segments;
    int out_offset = (blockIdx.y * gridDim.x + blockIdx.x) * (stride / num_sub_segments)
        + blockIdx.x % num_sub_segments * num_top_k;

    if (threadIdx.x == 0)
    {
        // We have to initialize active_count_per_batch for the following allClassNMS kernel.
        // Do it here to avoid to avoid an extra memset launch.
        if (blockIdx.x == 0)
        {
            active_count_per_batch[blockIdx.y] = 0;
        }
        active_count = 0;
    }
    __syncthreads();

    int first_index = segment * items;
    in += first_index;
    out += out_offset;
    out_indices += out_offset;

    int index_limit = items;
    uint32_t thread_items[ITEMS_PER_THREAD];
    int local_filtered = 0;

    // number of items whose score is >0 int he current thread
    int thread_active = 0;
    // in case <= top_k are active, offset where to write the thread items to in the output
    int thread_offset = 0;

    if (background_class_id != class_id)
    {
#pragma unroll
        for (int i = 0; i < ITEMS_PER_THREAD; ++i)
        {
            int offset = threadIdx.x + i * blockDim.x;
            thread_items[i] = 0;
            if (offset < index_limit)
            {
                thread_items[i] = in[offset];
            }
        }

        for (int i = 0; i < ITEMS_PER_THREAD; ++i)
        {
            if (__int_as_float(thread_items[i]) < threshold)
            {
                thread_items[i] = 0;

                // todo a bitmask + popc might be faster here
                int offset = threadIdx.x + i * blockDim.x;
                if (offset < index_limit)
                {
                    ++local_filtered;
                }
            }
            if (thread_items[i] > 0)
            {
                thread_active++;
            }
        }
#ifdef SSD_STABLE_TOPK
        BlockScan(temp_storage).ExclusiveSum(thread_active, thread_offset, active_count);
#else
        thread_offset = atomicAdd(&active_count, thread_active);
#endif
    }

    uint32_t select_mask = 0;
    uint32_t save_mask = 0;
    uint32_t save_bit = 0;

    if (threadIdx.x == 0)
    {
        selected_count = 0;
        old_selected_count = 0;
    }
    __syncthreads();

    if (threadIdx.x == 0)
    {
        active_counts_per_class[segment] = active_count;
    }

    // !!! experiment
    // all elements are filtered, nothing to do
    //   if (active_count == 0) {
    //       return;
    //   }

    // we have at maximum top_k elements. there's no need to filter those, store them directly as result.
    if (active_count <= num_top_k)
    {
        for (int i = 0; i < ITEMS_PER_THREAD; ++i)
        {
            if (thread_items[i] != 0)
            {
                out_indices[thread_offset] = threadIdx.x + i * blockDim.x + items * blockIdx.x;
                out[thread_offset] = thread_items[i];
                ++thread_offset;
            }
        }
        return;
    }

    // iterate over bits.
    // skip the first two bits,
    // * bit 31 is the sign bit. all values are positive
    // * bit 30 is only set for values >= 2, but the input consists only of values in the range of [0,1]
    const int skip_bits = 2;
    int selected;
    for (int bit = 31 - skip_bits; true; --bit)
    {
        __syncthreads();
        uint32_t bit_mask = select_mask | (1u << bit);

        uint32_t enabled = 0;
        for (int item = 0; item < ITEMS_PER_THREAD; ++item)
        {
            enabled |= (((thread_items[item] ^ bit_mask) & bit_mask) == 0) << item;
        }

        selected = __popc(enabled);
#ifdef SSD_STABLE_TOPK
        int offset;
        BlockScan(temp_storage).ExclusiveSum(selected, offset, prefix_op);
        if (threadIdx.x == 0)
        {
            selected_count = prefix_op.running_total;
        }
#else
        unsigned int offset = atomicAdd(&selected_count, selected);
#endif

        __syncthreads();
        int sc = selected_count;
        __syncthreads();

        if ((sc <= num_top_k && sc > 0) || (bit == 0 && sc > 0))
        {
            for (int item = 0; item < ITEMS_PER_THREAD; ++item)
            {
                if (enabled & (1u << item) && offset < num_top_k)
                {
                    selected_elements[offset]
                        = make_int2(thread_items[item], threadIdx.x + item * blockDim.x + items * blockIdx.x);
                    ++offset;
                    thread_items[item] = 0;
                }
            }
        }

        if (sc == num_top_k || bit == 0)
        {
            break;
        }
        else if (sc > num_top_k)
        {
            // There are too many bits in the current selection
            // Save the current state and go to the next bit
            // If there are not enough items left using the next bit
            // it's necessary to restart here with the current bit not set
            save_mask = bit_mask;
            save_bit = bit - 1;
            select_mask |= bit_mask;

            if (threadIdx.x == 0)
            {
                selected_count = old_selected_count;
#ifdef SSD_STABLE_TOPK
                prefix_op.running_total = old_selected_count;
#endif
            }
        }
        else
        {
            if (save_mask)
            {
                select_mask = save_mask;
                bit = save_bit;

                save_mask = 0;
            }
            if (threadIdx.x == 0)
            {
                old_selected_count = sc;
            }
        }
    }

    __syncthreads();

    // store data to global memory
    int sc = selected_count;
    for (int i = threadIdx.x; i < num_top_k; i += BLOCK_THREADS)
    {
        int2 selected_element = selected_elements[i];
        int out_element = i < sc ? selected_element.x : 0;
        out[i] = out_element;
        out_indices[i] = out_element > 0 ? selected_element.y : -1;
    }

    if (threadIdx.x == 0)
    {
        active_counts_per_class[segment] = num_top_k;
    }
}

} // namespace

template <typename T_SCORE>
ssdStatus_t topKScoresPerClass_gpu(hipStream_t stream, const int num, const int num_classes,
    const int num_preds_per_class, const int num_top_k, const int background_label_id, const float confidence_threshold,
    void* conf_scores_gpu, void* index_array_gpu, void* active_counts_gpu, void* active_counts_per_batch_gpu,
    void* temp_scores, void* temp_indices, void* temp_active_counts, size_t temp_storage_bytes, void* workspace,
    const int num_priors, const int num_dim, bool confSigmoid, const void* const* conf_data, const int num_layers,
    const int* feature_size, const int* num_anchors, const int* box_channels, const bool packed32NCHW)
{

#if SSD_RETINA_NET == 1

#if USE_CUB_SEGMENTED_SORT == 1

    int segments = num * num_classes;
    int items = num * num_classes * num_preds_per_class;

#if SSD_CUBLAS_CONF_PERMUTE == 1

    // scores data flow:
    // conf_data->conf_scores_gpu(first transpose/concat)->temp_scores(output scores)
    permuteConfDataFuseCubTopKPrepare(stream, num * num_classes * num_preds_per_class, num_classes, num_preds_per_class,
        num_layers, confSigmoid,
#if ENABLE_FUSED_TRANSPOSE == 1
        temp_scores, conf_scores_gpu,
#else
        conf_scores_gpu, temp_scores,
#endif
        // temp_scores,
        conf_data, active_counts_gpu, feature_size, num_anchors, packed32NCHW, temp_indices, background_label_id,
        confidence_threshold);
    // TODO move inside the fuse kernel
    hipMemsetAsync(active_counts_per_batch_gpu, 0, num * sizeof(int), stream);
#endif

#if ENABLE_FUSED_TRANSPOSE == 1 && SSD_CUBLAS_CONF_PERMUTE == 1
    {
        constexpr int BLOCK_SIZE = 256;
        int grid = div_up(segments, BLOCK_SIZE);

        get_cub_offsets_kernel<BLOCK_SIZE><<<grid, BLOCK_SIZE, 0, stream>>>(reinterpret_cast<int*>(active_counts_gpu),
            reinterpret_cast<int*>(active_counts_gpu), reinterpret_cast<int*>(temp_active_counts), items, segments,
            num_top_k);
    }

#else

    DEBUG_PRINTF("cub_top_k_prepare: items = %d, segments = %d, stride = %d\n", items, segments, items / segments);
    cub_top_k_prepare<T_SCORE, 1024><<<dim3(num_classes, num), 1024, 0, stream>>>(
        reinterpret_cast<T_SCORE*>(conf_scores_gpu), reinterpret_cast<T_SCORE*>(temp_scores),
        reinterpret_cast<int*>(temp_indices), reinterpret_cast<int*>(active_counts_gpu),
        reinterpret_cast<int*>(temp_active_counts), reinterpret_cast<int*>(active_counts_per_batch_gpu), items,
        segments, background_label_id, confidence_threshold);

#endif

    hipcub::DeviceSegmentedRadixSort::SortPairsDescending(workspace, temp_storage_bytes,
        reinterpret_cast<T_SCORE*>(temp_scores), reinterpret_cast<T_SCORE*>(conf_scores_gpu),
        reinterpret_cast<int*>(temp_indices), reinterpret_cast<int*>(index_array_gpu), items, segments,
        (int*) active_counts_gpu, (int*) temp_active_counts, 0, sizeof(T_SCORE) * 8, stream);

    get_active_counts<128><<<div_up(segments, 128), 128, 0, stream>>>(reinterpret_cast<int*>(temp_active_counts),
        reinterpret_cast<int*>(active_counts_gpu), items, segments, num_top_k);

#endif // USE_CUB_SEGMENTED_SORT

#else
    const int BLOCK_THREADS = TOPK_PER_CLASS_BLOCK_THREADS;

    using top_k_kernel = void (*)(int* in, int* out, int* out_indices, int* active_counts_gpu,
        int* active_counts_per_batch_gpu, int items, unsigned int num_top_k, int segments, int num_sub_segments,
        int background_class_id, float threshold);
    top_k_kernel top_k_kernels[] = {
        top_k_cuda_fused_prepare<1, BLOCK_THREADS>,
        top_k_cuda_fused_prepare<2, BLOCK_THREADS>,
        top_k_cuda_fused_prepare<3, BLOCK_THREADS>,
        top_k_cuda_fused_prepare<4, BLOCK_THREADS>,
        top_k_cuda_fused_prepare<5, BLOCK_THREADS>,
        top_k_cuda_fused_prepare<6, BLOCK_THREADS>,
        top_k_cuda_fused_prepare<7, BLOCK_THREADS>,
        top_k_cuda_fused_prepare<8, BLOCK_THREADS>,
        top_k_cuda_fused_prepare<9, BLOCK_THREADS>,
        top_k_cuda_fused_prepare<10, BLOCK_THREADS>,
        top_k_cuda_fused_prepare<11, BLOCK_THREADS>,
        top_k_cuda_fused_prepare<12, BLOCK_THREADS>,
        top_k_cuda_fused_prepare<13, BLOCK_THREADS>,
        top_k_cuda_fused_prepare<14, BLOCK_THREADS>,
        top_k_cuda_fused_prepare<15, BLOCK_THREADS>,
        top_k_cuda_fused_prepare<16, BLOCK_THREADS>,
        top_k_cuda_fused_prepare<17, BLOCK_THREADS>,
        top_k_cuda_fused_prepare<18, BLOCK_THREADS>,
        top_k_cuda_fused_prepare<19, BLOCK_THREADS>,
        top_k_cuda_fused_prepare<20, BLOCK_THREADS>,
        top_k_cuda_fused_prepare<21, BLOCK_THREADS>,
        top_k_cuda_fused_prepare<22, BLOCK_THREADS>,
        top_k_cuda_fused_prepare<23, BLOCK_THREADS>,
        top_k_cuda_fused_prepare<24, BLOCK_THREADS>,
        top_k_cuda_fused_prepare<25, BLOCK_THREADS>,
        top_k_cuda_fused_prepare<26, BLOCK_THREADS>,
        top_k_cuda_fused_prepare<27, BLOCK_THREADS>,
        top_k_cuda_fused_prepare<28, BLOCK_THREADS>,
        top_k_cuda_fused_prepare<29, BLOCK_THREADS>,
        top_k_cuda_fused_prepare<30, BLOCK_THREADS>,
        top_k_cuda_fused_prepare<31, BLOCK_THREADS>,
        top_k_cuda_fused_prepare<32, BLOCK_THREADS>,
    };

    const int num_segments = num * num_classes;

    uint32_t smem_size = num_top_k * (sizeof(int) + sizeof(uint32_t));

    // TODO implement multi-stage topk
    int kernel_index = (num_preds_per_class + BLOCK_THREADS - 1) / BLOCK_THREADS;
    dim3 block(BLOCK_THREADS);

    bool do_n_pass = (kernel_index >= 32);

    DEBUG_PRINTF("num_preds_per_class = %d\n", num_preds_per_class);

    // do_n_pass = false;  // debug resnet34

    void* out_scores = (do_n_pass) ? temp_scores : conf_scores_gpu;
    void* out_indices = (do_n_pass) ? temp_indices : index_array_gpu;
    void* out_active_counts = (do_n_pass) ? temp_active_counts : active_counts_gpu;

    // number of segments each class is split into
    int num_sub_segments = 1; //&& num_preds_per_class % num_sub_segments
    while (kernel_index >= 32)
    {
        // introduce additional step
        num_sub_segments += 1;
        int items_per_sub_segment = div_up(num_preds_per_class, num_sub_segments);
        kernel_index = div_up(items_per_sub_segment, TOPK_PER_CLASS_BLOCK_THREADS);
    }

    //!!! hard_code for now:
    if (do_n_pass)
    {
        // num_sub_segments = 2;  // debug resnet34
        num_sub_segments = 9;
        int items_per_sub_segment = num_preds_per_class / num_sub_segments;
        kernel_index = div_up(items_per_sub_segment, TOPK_PER_CLASS_BLOCK_THREADS);
        DEBUG_PRINTF("Using kernel #%d\n", kernel_index);
    }

    assert(num_preds_per_class % num_sub_segments == 0);

    dim3 grid(num_classes * num_sub_segments, num);

    assert(kernel_index < 32);

    DEBUG_PRINTF("top_k Per Class\n");

    if (do_n_pass)
    {
        hipMemcpyAsync(
            out_scores, conf_scores_gpu, num_preds_per_class * num_segments, hipMemcpyDeviceToDevice, stream);
    }

    top_k_kernels[kernel_index]<<<grid, block, smem_size, stream>>>((int*) (conf_scores_gpu), (int*) (out_scores),
        (int*) out_indices, (int*) out_active_counts, (int*) active_counts_per_batch_gpu,
        num_preds_per_class / num_sub_segments, num_top_k, num_classes * num_sub_segments, num_sub_segments,
        background_label_id, confidence_threshold);

    if (do_n_pass)
    {
        top_k_multi_pass((int*) (out_scores), (int*) out_indices, (int*) (conf_scores_gpu), (int*) index_array_gpu,
            (int*) out_active_counts, (int*) active_counts_per_batch_gpu, num_top_k * num_sub_segments,
            num_preds_per_class, num_top_k, num_sub_segments, num_segments, stream);
    }

    blockSort<T_SCORE>((const T_SCORE*) (out_scores), (T_SCORE*) (conf_scores_gpu), (const int*) (out_indices),
        (int*) (index_array_gpu), (int*) out_active_counts, num_top_k, num_preds_per_class, num_segments, stream);

#endif // #if SSD_RETINA_NET

    CSC(hipGetLastError(), STATUS_FAILURE);
    return STATUS_SUCCESS;
}

// sortScoresPerClass LAUNCH CONFIG {{{
typedef ssdStatus_t (*tkspcFunc)(hipStream_t, const int, const int, const int, const int, const int, const float,
    void*, void*, void*, void*, void*, void*, void*, size_t, void*, const int, const int, bool, const void* const*,
    const int, const int*, const int*, const int*, const bool);
struct tkspcLaunchConfig
{
    DType_t t_score;
    tkspcFunc function;

    tkspcLaunchConfig(DType_t t_score)
        : t_score(t_score)
    {
    }
    tkspcLaunchConfig(DType_t t_score, tkspcFunc function)
        : t_score(t_score)
        , function(function)
    {
    }
    bool operator==(const tkspcLaunchConfig& other)
    {
        return t_score == other.t_score;
    }
};

static std::vector<tkspcLaunchConfig> tkspcFuncVec;
bool tkspcInit()
{
    tkspcFuncVec.push_back(tkspcLaunchConfig(DataType::kFLOAT, topKScoresPerClass_gpu<float>));
    return true;
}

static bool initialized = tkspcInit();
//}}}

ssdStatus_t topKScoresPerClass(hipStream_t stream, const int num, const int num_classes, const int num_preds_per_class,
    const int num_top_k, const int background_label_id, const float confidence_threshold, const DType_t DT_SCORE,
    void* conf_scores_gpu, void* index_array_gpu, void* active_count_per_class, void* active_count_per_batch,
    void* temp_scores, void* temp_indices, void* temp_active_counts, size_t temp_storage_bytes, void* workspace,
    const int num_priors, const int num_dim, bool confSigmoid, const void* const* conf_data, const int num_layers,
    const int* feature_size, const int* num_anchors, const int* box_channels, const bool packed32NCHW)
{
    tkspcLaunchConfig lc = tkspcLaunchConfig(DT_SCORE);
    for (unsigned i = 0; i < tkspcFuncVec.size(); ++i)
    {
        if (lc == tkspcFuncVec[i])
        {
            DEBUG_PRINTF("sortScoresPerClass kernel %d\n", i);
            return tkspcFuncVec[i].function(stream, num, num_classes, num_preds_per_class, num_top_k,
                background_label_id, confidence_threshold, conf_scores_gpu, index_array_gpu, active_count_per_class,
                active_count_per_batch, temp_scores, temp_indices, temp_active_counts, temp_storage_bytes, workspace,
                num_priors, num_dim, confSigmoid, conf_data, num_layers, feature_size, num_anchors, box_channels,
                packed32NCHW);
        }
    }
    return STATUS_BAD_PARAM;
}

size_t topKScoresPerClassWorkspaceSize(const int num, const int num_classes, const int num_preds_per_class,
    const int num_top_k, const DType_t DT_CONF, size_t& temp_active_counts_size, size_t& temp_sort_scores_size,
    size_t& temp_sort_indicies_size)
{
    int data_type_size = (DT_CONF == DataType::kFLOAT)
        ? sizeof(float)
        : (DT_CONF == DataType::kHALF) ? sizeof(uint16_t) : sizeof(uint8_t);

    int kernel_index = div_up(num_preds_per_class, TOPK_PER_CLASS_BLOCK_THREADS);
    temp_active_counts_size = temp_sort_scores_size = temp_sort_indicies_size = 0;

    // comment for debug
    // if (kernel_index < 32) return 0;

    // number of segments each class is split into
    int num_sub_segments = 1;

    size_t temp_cub_storage_bytes = 0;

#if USE_CUB_SEGMENTED_SORT == 1

    hipcub::DeviceSegmentedRadixSort::SortPairsDescending(nullptr, temp_cub_storage_bytes, (float*) nullptr,
        (float*) nullptr, (int*) nullptr, (int*) nullptr, num * num_classes * num_preds_per_class, num * num_classes,
        (int*) nullptr, (int*) nullptr, 0, sizeof(int) * 8, 0);
#else
    while (kernel_index >= 32)
    {
        // introduce additional step
        num_sub_segments += 1;
        int items_per_sub_segment = div_up(num_preds_per_class, num_sub_segments);
        kernel_index = div_up(items_per_sub_segment, TOPK_PER_CLASS_BLOCK_THREADS);
    }
    //!!! hard_code for now:
    num_sub_segments = 10; // debug resnet34
                           // num_sub_segments = 9;
#endif

    temp_active_counts_size = num * num_classes * num_sub_segments * sizeof(int);
    size_t num0 = (size_t) num * (size_t) num_classes * (size_t) num_sub_segments * (size_t) num_top_k;
    size_t num1 = (size_t) num * (size_t) num_classes * (size_t) num_preds_per_class;
    temp_sort_scores_size = std::max(num0, num1) * data_type_size;
    temp_sort_indicies_size = std::max(num0, num1) * sizeof(int);
    DEBUG_PRINTF("Temp storage calculation: num_segments required = %d\n", num_sub_segments);
    DEBUG_PRINTF("Active counts temp arrays size = %d\n", temp_active_counts_size);
    DEBUG_PRINTF("Sorting temp arrays size = %d\n", temp_sort_scores_size);

    return temp_active_counts_size + temp_sort_scores_size + temp_sort_indicies_size + temp_cub_storage_bytes;
}

} // namespace plugin
} // namespace nvinfer1
