#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <stdint.h>

#include <hipcub/hipcub.hpp>

#include "nms_common.h"
#include "ssdOptMacros.h"
#include "topK.h"

constexpr int TOPK_BLOCK_THREADS = 512;

namespace
{

/*************************************************************************************************/

struct BlockPrefixCallbackOp
{
    // Running prefix
    int running_total;
    // Constructor
    __device__ BlockPrefixCallbackOp(int running_total)
        : running_total(running_total)
    {
    }
    // Callback operator to be entered by the first warp of threads in the block.
    // Thread-0 is responsible for returning a value for seeding the block-wide scan.
    __device__ int operator()(int block_aggregate)
    {
        int old_prefix = running_total;
        running_total += block_aggregate;
        return old_prefix;
    }
};

/*************************************************************************************************/

const int SEGMENTED_SCAN_BLOCK_THREADS = 128;

__global__ void segmented_scan_kernel(int* in, int* out, int* aggregate, int* max, int segments)
{
    // Specialize BlockScan type for our thread block
    // can be in-place
    typedef hipcub::BlockScan<int, SEGMENTED_SCAN_BLOCK_THREADS> BlockScan;
    __shared__ typename BlockScan::TempStorage temp_storage;
    // Initialize running total
    BlockPrefixCallbackOp prefix_op(0);

    const int offset = blockIdx.x * segments;
    in += offset;
    out += offset;

    int finish = div_up(segments, SEGMENTED_SCAN_BLOCK_THREADS) * SEGMENTED_SCAN_BLOCK_THREADS;
    for (int idx = threadIdx.x; idx < finish; idx += SEGMENTED_SCAN_BLOCK_THREADS)
    {
        int thread_count = (idx < segments) ? in[idx] : 0;

        int thread_offset;
        BlockScan(temp_storage).ExclusiveSum(thread_count, thread_offset, prefix_op);

        __syncthreads();

        if (idx < segments)
            out[idx] = thread_offset;
    }

    if (threadIdx.x == 0)
    {
        aggregate[blockIdx.x] = prefix_op.running_total;
        // TODO: we can detect num_items based on the outcome of reduction
        /*     if (max != NULL) {
              atomicMax(max, prefix_op.running_total);
            } */
    }
}

/*************************************************************************************************/

__global__ void compact_segments_kernel(const int* in, const int* in_indices, int* out, int* out_indices,
    int* active_count_scan, int* active_count_per_batch, int items, int stride)
{
    // Each batch contain "segment" pieces
    // The kernel compacts segments into the beginning of each batch
    // items = max number of items per batch
    // items % segment == 0
    //  number of segments per batch
    const int num_segment = gridDim.x;
    const int batch_id = blockIdx.y;

    active_count_scan += batch_id * num_segment;

    const int segment_id = blockIdx.x;
    int segment_size = items / num_segment;

    int offset_in = batch_id * stride + segment_id * segment_size;
    int offset_out = batch_id * stride + active_count_scan[segment_id];

    int items_per_segment = (segment_id < num_segment - 1)
        ? active_count_scan[segment_id + 1] - active_count_scan[segment_id]
        : active_count_per_batch[batch_id] - active_count_scan[segment_id];

    for (int idx = threadIdx.x; idx < items_per_segment; idx += blockDim.x)
    {
        out[offset_out + idx] = in[offset_in + idx];
        out_indices[offset_out + idx] = in_indices[offset_in + idx];
    }
}

/*************************************************************************************************/

template <int ITEMS_PER_THREAD, int BLOCK_THREADS>
__global__ void top_k_cuda(int* in, int* in_indices, int* out, int* out_indices, int* active_count,
    int* active_count_per_batch, int items, int stride, unsigned int num_top_k)
{
    extern __shared__ uint32_t dynamic_memory[];
    uint32_t* selected_items = dynamic_memory;
    int32_t* selected_indices = reinterpret_cast<int32_t*>(selected_items + num_top_k);
    __shared__ unsigned int selected_count;
    unsigned int old_selected_count;

// Specialize BlockScan type for our thread block
#ifdef SSD_STABLE_TOPK
    typedef hipcub::BlockScan<int, BLOCK_THREADS> BlockScan;
    __shared__ typename BlockScan::TempStorage temp_storage;
    // Initialize running total
    BlockPrefixCallbackOp prefix_op(0);
#endif

    int batch = blockIdx.x;
    int first_index = batch * stride;

    // segments per batch
    int num_segments = gridDim.y;
    int items_per_segment = div_up(items, num_segments);
    int segment_items_offset = blockIdx.y * items_per_segment;
    items = active_count_per_batch[batch];
    if (items < segment_items_offset)
    {
        items = 0;
    }

    items = min(items_per_segment, items - segment_items_offset);
    // items = (items < 0)? -items : items;
    int second_index = first_index + blockIdx.y * num_top_k;
    first_index += segment_items_offset;

    in += first_index;
    in_indices += first_index;

    out += second_index;
    out_indices += second_index;

    // Feed input
    uint32_t thread_items[ITEMS_PER_THREAD];
    int32_t thread_indices[ITEMS_PER_THREAD];

    for (int i = 0; i < ITEMS_PER_THREAD; ++i)
    {
        int offset = threadIdx.x + i * blockDim.x;
        if (offset < items)
        {
            thread_items[i] = in[offset];
            thread_indices[i] = in_indices[offset];
        }
        else
        {
            thread_items[i] = 0;
            thread_indices[i] = -1;
        }
    }

    if (items <= num_top_k)
    {
        if (threadIdx.x == 0)
        {
            if (gridDim.y == 1)
            {
                active_count_per_batch[batch] = items;
            }
            else
            {
                // this is preliminary step, so we need to populate active_count
                active_count[blockIdx.x * num_segments + blockIdx.y] = items;
            }
        }

        // we know that the results are compact, so we can bail out early.
        for (int i = 0; i < ITEMS_PER_THREAD; ++i)
        {
            int offset = threadIdx.x + i * blockDim.x;
            if (offset < num_top_k)
            {
                out[offset] = thread_items[i];
                out_indices[offset] = thread_indices[i];
            }
            else
            {
                return;
            }
        }
    }

    uint32_t select_mask = 0;
    uint32_t save_mask = 0;
    uint32_t save_bit = 0;

    if (threadIdx.x == 0)
    {
        selected_count = 0;
        old_selected_count = 0;
    }

#define MTA_D 0

    // iterate over bits.
    // skip the first two bits,
    // * bit 31 is the sign bit. all values are positive
    // * bit 30 is only set for values >= 2, but the input consists only of values in the range of [0,1]
    const int skip_bits = 0;
    int selected = 0;
    for (int bit = 31 - skip_bits; true; --bit)
    {
        __syncthreads();
        uint32_t bit_mask = select_mask | (1u << bit);

        uint32_t enabled = 0;
        for (int item = 0; item < ITEMS_PER_THREAD; ++item)
        {
            enabled |= (((thread_items[item] ^ bit_mask) & bit_mask) == 0) << item;
        }

        selected = __popc(enabled);
#ifdef SSD_STABLE_TOPK
        int offset;
        BlockScan(temp_storage).ExclusiveSum(selected, offset, prefix_op);
        if (threadIdx.x == 0)
        {
            selected_count = prefix_op.running_total;
        }
#else
        unsigned int offset = atomicAdd(&selected_count, selected);
#endif

        __syncthreads();
        int sc = selected_count;
        __syncthreads();

        if ((sc <= num_top_k && sc > 0) || (bit == 0 && sc > 0))
        {
            for (int item = 0; item < ITEMS_PER_THREAD; ++item)
            {
                if (enabled & (1u << item) && offset < num_top_k)
                {
                    selected_items[offset] = thread_items[item];
                    selected_indices[offset] = thread_indices[item];
                    ++offset;
                    thread_items[item] = 0;
                }
            }
        }

        if (sc == num_top_k || bit == 0)
        {
            break;
        }
        else if (sc > num_top_k)
        {
            // There are too many bits in the current selection
            // Save the current state and go to the next bit
            // If there are not enough items left using the next bit
            // it's necessary to restart here with the current bit not set
            save_mask = bit_mask;
            save_bit = bit - 1;
            select_mask |= bit_mask;

            if (threadIdx.x == 0)
            {
                selected_count = old_selected_count;
#ifdef SSD_STABLE_TOPK
                prefix_op.running_total = old_selected_count;
#endif
            }
        }
        else
        {
            if (save_mask)
            {
                select_mask = save_mask;
                bit = save_bit;

                save_mask = 0;
            }
            if (threadIdx.x == 0)
            {
                old_selected_count = sc;
            }
        }
    }

    __syncthreads();

    // store data to global memory
    int sc = selected_count;
    for (int i = threadIdx.x; i < num_top_k; i += blockDim.x)
    {
        out[i] = (i < sc) ? selected_items[i] : 1;
        out_indices[i] = (i < sc && selected_items[0] > 0) ? selected_indices[i] : -1;
    }

    if (threadIdx.x == 0)
    {
        if (gridDim.y == 1)
        {
            active_count_per_batch[batch] = num_top_k;
        }
        else
        {
            active_count[batch * num_segments + blockIdx.y] = num_top_k;
        }
    }
}

} // end namespace

/*************************************************************************************************/

void segmented_scan(int* active_count_in, int* active_count_out, int* active_count_per_batch, int num_images,
    int num_classes, hipStream_t stream)
{

    // recomputes active_count_per_batch
    segmented_scan_kernel<<<num_images, SEGMENTED_SCAN_BLOCK_THREADS, 0, stream>>>(
        active_count_in, active_count_out, active_count_per_batch, NULL, num_classes);
}

/*************************************************************************************************/

void compact_segments(void* input_output_scores, void* input_output_indices, void* temp_scores, void* temp_indices,
    int* active_count, int* active_count_per_batch, int num_classes, int num_images, int num_items_per_image,
    int image_stride, hipStream_t stream)
{

    compact_segments_kernel<<<dim3(num_classes, num_images, 1), 128, 0, stream>>>((int*) input_output_scores,
        (int*) input_output_indices, (int*) temp_scores, (int*) temp_indices, (int*) active_count,
        (int*) active_count_per_batch, num_items_per_image, image_stride);
}

/*************************************************************************************************/

using top_k_kernel = void (*)(int* in, int* in_indices, int* out, int* out_indices, int* active_count,
    int* active_count_per_batch, int items, int stride_items, unsigned int num_top_k);

/*************************************************************************************************/

template <int BLOCK_THREADS>
void top_k(top_k_kernel* kernel, int kernel_index, void* input_output_scores, void* input_output_indices,
    void* temp_scores, void* temp_indices, int* active_count, int* active_count_per_batch, int num_items_per_image,
    int image_stride, int num_top_k, int num_classes, int num_images, int num_segments, hipStream_t& stream)
{
    // num_images - barch size
    // num_classes - here number of segments per batch(image)
    // num_items_per_image - total items per single batch

    DEBUG_PRINTF("top_k: num_segments = %d, num_classes = %d\n", num_segments, num_classes);

    if (num_items_per_image % num_classes)
    {
        DEBUG_PRINTF("topK Warning: num_items_per_image % num_classes != 0\n");
    }

    assert(num_items_per_image % num_classes == 0);

    // recomputes active_count_per_batch
    segmented_scan(active_count, active_count, active_count_per_batch, num_images, num_classes, stream);

    compact_segments(input_output_scores, input_output_indices, temp_scores, temp_indices, active_count,
        active_count_per_batch, num_classes, num_images, num_items_per_image, image_stride, stream);

    uint32_t smem_size = num_top_k * (sizeof(int) + sizeof(uint32_t));

    kernel[kernel_index]<<<dim3(num_images, num_segments, 1), BLOCK_THREADS, smem_size, stream>>>((int*) (temp_scores),
        (int*) temp_indices, (int*) (input_output_scores), (int*) input_output_indices, (int*) active_count,
        (int*) active_count_per_batch, num_items_per_image, image_stride, num_top_k);
}

/*************************************************************************************************/

template <int BLOCK_THREADS>
void top_k_n_pass(top_k_kernel* top_k_kernels, void* input_output_scores, void* input_output_indices, void* temp_scores,
    void* temp_indices, int* active_count, int* active_count_per_batch, int num_items_per_image, int image_stride,
    int num_top_k, int num_classes, int num_images, hipStream_t& stream)
{

    // num_images - barch size
    // num_classes - here initial number of segments per batch
    // num_items_per_image - total items per single batch

    // printf("top_k_n_pass: num_top_k = %d, num_classes = %d, num_images = %d, num_items_per_image = %d, image_stride =
    // %d\n", num_top_k, num_classes, num_images, num_items_per_image, image_stride);

    int kernel_index = div_up(num_items_per_image, BLOCK_THREADS);

    int num_segments = 1;
    while (kernel_index >= 32)
    {
        // introduce additional step
        num_segments += 1;
        int items_per_sub_segment = div_up(num_items_per_image, num_segments);
        kernel_index = (items_per_sub_segment + BLOCK_THREADS - 1) / BLOCK_THREADS;
    }
    // num_segments = (num_segments == 8)? 1 : num_segments;

    top_k<BLOCK_THREADS>(top_k_kernels, kernel_index, input_output_scores, input_output_indices, temp_scores,
        temp_indices, active_count, active_count_per_batch, num_items_per_image, image_stride, num_top_k, num_classes,
        num_images, num_segments, stream);
    if (num_segments > 1)
    {
        int num_items_per_image_stage_next = num_segments * num_top_k;
        kernel_index = div_up(num_items_per_image_stage_next, BLOCK_THREADS);
        if (kernel_index >= 32)
        {
            top_k_n_pass<BLOCK_THREADS>(top_k_kernels, input_output_scores, input_output_indices, temp_scores,
                temp_indices, active_count, active_count_per_batch, num_items_per_image_stage_next, image_stride,
                num_top_k, num_segments, num_images, stream);
        }
        else
        {
            top_k<BLOCK_THREADS>(top_k_kernels, kernel_index, input_output_scores, input_output_indices, temp_scores,
                temp_indices, active_count, active_count_per_batch, num_items_per_image_stage_next, image_stride,
                num_top_k, num_segments, num_images, 1, stream);
        }
    }
}

/*************************************************************************************************/

void top_k_multi_pass(void* input_output_scores, void* input_output_indices, void* temp_scores, void* temp_indices,
    int* active_count, int* active_count_per_batch, int num_items_per_image, int image_stride, int num_top_k,
    int num_classes, int num_images, hipStream_t& stream)
{

    // num_images - barch size
    // num_classes - here initial number of segments per batch
    // num_items_per_image - total items per single batch
    // active_count - counts per segment
    // active_count_per_batch - counts per batch, aggregated over the segments
    // IMPORTANT! input_output_scores/input_output_indices will be overwritten by output

    constexpr int BLOCK_THREADS = TOPK_BLOCK_THREADS;

    top_k_kernel top_k_kernels[] = {
        top_k_cuda<1, BLOCK_THREADS>,
        top_k_cuda<2, BLOCK_THREADS>,
        top_k_cuda<3, BLOCK_THREADS>,
        top_k_cuda<4, BLOCK_THREADS>,
        top_k_cuda<5, BLOCK_THREADS>,
        top_k_cuda<6, BLOCK_THREADS>,
        top_k_cuda<7, BLOCK_THREADS>,
        top_k_cuda<8, BLOCK_THREADS>,
        top_k_cuda<9, BLOCK_THREADS>,
        top_k_cuda<10, BLOCK_THREADS>,
        top_k_cuda<11, BLOCK_THREADS>,
        top_k_cuda<12, BLOCK_THREADS>,
        top_k_cuda<13, BLOCK_THREADS>,
        top_k_cuda<14, BLOCK_THREADS>,
        top_k_cuda<15, BLOCK_THREADS>,
        top_k_cuda<16, BLOCK_THREADS>,
        top_k_cuda<17, BLOCK_THREADS>,
        top_k_cuda<18, BLOCK_THREADS>,
        top_k_cuda<19, BLOCK_THREADS>,
        top_k_cuda<20, BLOCK_THREADS>,
        top_k_cuda<21, BLOCK_THREADS>,
        top_k_cuda<22, BLOCK_THREADS>,
        top_k_cuda<23, BLOCK_THREADS>,
        top_k_cuda<24, BLOCK_THREADS>,
        top_k_cuda<25, BLOCK_THREADS>,
        top_k_cuda<26, BLOCK_THREADS>,
        top_k_cuda<27, BLOCK_THREADS>,
        top_k_cuda<28, BLOCK_THREADS>,
        top_k_cuda<29, BLOCK_THREADS>,
        top_k_cuda<30, BLOCK_THREADS>,
        top_k_cuda<31, BLOCK_THREADS>,
        top_k_cuda<32, BLOCK_THREADS>,
    };

    top_k_n_pass<BLOCK_THREADS>(top_k_kernels, input_output_scores, input_output_indices, temp_scores, temp_indices,
        (int*) active_count, (int*) active_count_per_batch, num_items_per_image, image_stride, num_top_k, num_classes,
        num_images, stream);
}
