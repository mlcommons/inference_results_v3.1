/*
 * Copyright (c) 2023, NVIDIA CORPORATION. All rights reserved.
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cstdio>

#include <hip/hip_fp16.h>
#include <cuda_pipeline_primitives.h>

#include "embedding_gather_launch.hpp"
#include "embedding_gather_kernel.cuh"
#include "launch_host_init.hpp"

int run_mega_embedding_gather
(
  hipStream_t stream,
  /* the indices.
   * current always using int32_t type
   */
  const int*   sparse_input,
  const int*   index_remap,
  const int*   index_hotnesses,
  const int*   index_offsets,
  
      IOType   io_type,
  
  /* the tables
   * dense_input is also viewed as a separate table
   */
  const void*  dense_input,
  const void*  mega_table,
  const void*  mega_table_host,

        void*  output,

        int    batch_size,
        int    embed_dim,           /*the table width(element)*/
        int    embed_feature_total, /*the number of categorical features*/
        int    embed_hotness_total,
        int    embed_rows_gpu,

  const float* scales,
  const float* scales_inv
) {
  
  using IndexType = int;
  const int BYTES_PER_INDEX_ELT = sizeof(IndexType);

  if      (io_type == IOType::FLOAT)
  {
    using EmbedType = float;

    const int BYTES_PER_EMBED_ELT = sizeof(float);
    const int NUM_SAMPLES_PER_CTA = 2;
    const int BYTES_LDST_EMBED    = 16;

    LAUNCH_HOST_INIT; PRINT_HOST_INIT;
    
    if (index_remap == nullptr) {
      LAUNCH_WO_INDEX_REMAP;
    } else {
      LAUNCH_INDEX_REMAP;
    }
    
  }
  else if (io_type == IOType::HALF)
  {
    using EmbedType = half;

    const int BYTES_PER_EMBED_ELT = sizeof(half);
    const int NUM_SAMPLES_PER_CTA = 4;
    const int BYTES_LDST_EMBED    = 16;

    LAUNCH_HOST_INIT; PRINT_HOST_INIT;

    if (index_remap == nullptr) {
      LAUNCH_WO_INDEX_REMAP;
    } else {
      LAUNCH_INDEX_REMAP;
    }
    
  }
  else if (io_type == IOType::INT8)
  {
    using EmbedType = int8_t;

    const int BYTES_PER_EMBED_ELT = sizeof(int8_t);
    const int NUM_SAMPLES_PER_CTA = 8;
    const int BYTES_LDST_EMBED    = 16;

    // const int NUM_SAMPLES_PER_CTA = 4;
    // const int BYTES_LDST_EMBED    = 8;

    // const int NUM_SAMPLES_PER_CTA = 2;
    // const int BYTES_LDST_EMBED    = 4;
    
    LAUNCH_HOST_INIT; PRINT_HOST_INIT;

    assert(scales     != nullptr);
    assert(scales_inv != nullptr);
    
    if (index_remap == nullptr) {
      LAUNCH_WO_INDEX_REMAP;
    } else {
      LAUNCH_INDEX_REMAP;
    }
    
  }
  
  return 0;
}