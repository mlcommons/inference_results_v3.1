#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "unet3d_sw.cuh"
#include <hip/hip_cooperative_groups.h>

namespace cg = cooperative_groups;
namespace lwis
{

__global__ void UNet3DKiTS19SliceKernelI8Linear(const int8_t* __restrict__ d_in, int8_t* __restrict__ d_out, const UNet3DParams p)
{
    const int d = blockIdx.x;
    const int h = blockIdx.y;
    const int w = threadIdx.x;

    int slice_id = blockIdx.z;
    int slice_offset_d = p.slice_to_off_d[slice_id];
    int slice_offset_h = p.slice_to_off_h[slice_id];
    int slice_offset_w = p.slice_to_off_w[slice_id];

    if (d < p.roi_dhw && h < p.roi_dhw && w < p.roi_dhw && slice_id < p.actual_num_slices)
    {
        d_out[(slice_id * p.roi_size) + (p.roi_dhw * p.roi_dhw * d + p.roi_dhw * h + w)]
            = d_in[p.image_w * p.image_h * (slice_offset_d + d) + p.image_w * (slice_offset_h + h)
                + (slice_offset_w + w)];
    }
}

// This impl assumes there's no race condition in read-modify-write of d_out
__global__ void UNet3DKiTS19PatchKernelNoOverlap(const __half* __restrict__ d_in, __half* __restrict__ d_out, const UNet3DParams p)
{
    const int d = blockIdx.x;
    const int h = blockIdx.y;
    const int w = threadIdx.x;

    const int slice_id = blockIdx.z;

    const int slice_offset_d = p.slice_to_off_d[slice_id];
    const int slice_offset_h = p.slice_to_off_h[slice_id];
    const int slice_offset_w = p.slice_to_off_w[slice_id];

    if (d < p.roi_dhw && h < p.roi_dhw && w < p.roi_dhw && slice_id < p.actual_num_slices)
    {
        #pragma unroll
        for (int c = 0; c < p.out_ch; ++c)
        {
            d_out[p.image_h * p.image_w * (slice_offset_d + d) + p.image_w * (slice_offset_h + h) + (slice_offset_w + w)
                  + p.image_size * c]
                += d_in[slice_id * p.roi_size * p.out_ch
                        + p.roi_dhw * p.roi_dhw * d + p.roi_dhw * h + w + p.roi_size * c]
                * ((__half*)p.patches[slice_id])[p.roi_dhw * p.roi_dhw * d + p.roi_dhw * h + w];
        }
    }
}

// This impl uses Cooperative Group and performs device-wide sync, to address race condition on d_out
__global__ void UNet3DKiTS19PatchKernelOverlapCG(const __half* __restrict__ d_in, __half* __restrict__ d_out, const UNet3DParams p)
{
    const int total_d = p.roi_dhw;
    const int total_h = p.roi_dhw;
    const int total_w = p.roi_dhw;
    const int d_stride = gridDim.x;
    const int h_stride = gridDim.y;
    const int w_stride = blockDim.x;

    int num_slices = p.actual_num_slices;
    for (int slice_id = 0; slice_id < num_slices; ++slice_id)
    {
        const int slice_offset_d = p.slice_to_off_d[slice_id];
        const int slice_offset_h = p.slice_to_off_h[slice_id];
        const int slice_offset_w = p.slice_to_off_w[slice_id];

        for (int dd = 0; dd < total_d; dd += d_stride)
        {
            int d = dd + blockIdx.x;
            for (int hh = 0; hh < total_h; hh += h_stride)
            {
                int h = hh + blockIdx.y;
                for (int ww = 0; ww < total_w; ww += w_stride)
                {
                    int w = ww + threadIdx.x;
                    if (d < total_d && h < total_h && w < total_w)
                    {
                        #pragma unroll
                        for (int c = 0; c < p.out_ch; ++c)
                        {
                            d_out[p.image_h * p.image_w * (slice_offset_d + d) + p.image_w * (slice_offset_h + h) + (slice_offset_w + w)
                                + p.image_size * c]
                                += d_in[slice_id * p.roi_size * p.out_ch
                                        + p.roi_dhw * p.roi_dhw * d + p.roi_dhw * h + w + p.roi_size * c]
                                * ((__half*)p.patches[slice_id])[p.roi_dhw * p.roi_dhw * d + p.roi_dhw * h + w];
                        }
                    }
                }
            }
        }
        cg::this_grid().sync();
    }
}

__global__ void UNet3DKiTS19ArgMaxKernel(
    const __half* __restrict__ d_in, int8_t* __restrict__ d_out, const UNet3DParams p)
{
    const int d = blockIdx.x;
    const int h = blockIdx.y;
    const int w = threadIdx.x;

    __half a = d_in[p.image_h * p.image_w * d + p.image_w * h + w];
    __half b = d_in[p.image_h * p.image_w * d + p.image_w * h + w + p.image_size];
    __half c = d_in[p.image_h * p.image_w * d + p.image_w * h + w + 2 * p.image_size];
    __half m = b;
    uint8_t l = 1;
    if (a > b)
    {
        m = a;
        l = 0;
    }
    if (d < p.image_d && h < p.image_h && w < p.image_w)
    {
        d_out[p.image_h * p.image_w * d + p.image_w * h + w] = m > c ? l : 2;
    }
}

void UNet3DKiTS19SliceKernelI8Linear_wrapper(void* d_in, void* d_out, const UNet3DParams& p, const hipStream_t stream = 0, const int deviceId = 0)
{
    hipSetDevice(deviceId);
    // for slicing
    dim3 dimBlock_slice(p.roi_dhw, 1, 1);
    dim3 dimGrid_slice(p.roi_dhw, p.roi_dhw, p.actual_num_slices);    
    UNet3DKiTS19SliceKernelI8Linear<<<dimGrid_slice, dimBlock_slice, 0, stream>>>(
        static_cast<int8_t*>(d_in), static_cast<int8_t*>(d_out), p);
}

void UNet3DKiTS19PatchKernelNoOverlap_wrapper(void* d_in, void* d_out, const UNet3DParams& p, const hipStream_t stream = 0, const int deviceId = 0)
{
    hipSetDevice(deviceId);
    // for Gaussian patching & accumulating
    dim3 dimBlock_patch(p.roi_dhw, 1, 1);
    dim3 dimGrid_patch(p.roi_dhw, p.roi_dhw, p.actual_num_slices);
    UNet3DKiTS19PatchKernelNoOverlap<<<dimGrid_patch, dimBlock_patch, 0, stream>>>(static_cast<__half*>(d_in), static_cast<__half*>(d_out), p);
}

// Using Cooperative Group, and device-wide sync
void UNet3DKiTS19PatchKernelOverlapCG_wrapper(void* d_in, void* d_out, const UNet3DParams& p, const hipStream_t stream = 0, const int deviceId = 0)
{
    hipSetDevice(deviceId);
    
    int BLOCKS;
    int THREADS;
    hipOccupancyMaxPotentialBlockSize(&BLOCKS, &THREADS, UNet3DKiTS19PatchKernelOverlapCG, 0, p.roi_dhw);

    int GridY = BLOCKS >= p.roi_dhw ? p.roi_dhw : BLOCKS;
    int GridX = BLOCKS < p.roi_dhw ? 1 : BLOCKS / GridY;

    void* kernel_args[] = { &d_in, &d_out, (void*)&p };
    // for Gaussian patching & accumulating
    dim3 dimBlock_patch(THREADS, 1, 1);
    dim3 dimGrid_patch(GridX, GridY, 1);

    hipLaunchCooperativeKernel((void*)(UNet3DKiTS19PatchKernelOverlapCG), dimGrid_patch, dimBlock_patch, kernel_args, 0, stream);
}

// Using CPU implicit sync; launches no-overlap kernel one by one
void UNet3DKiTS19PatchKernelOverlapImplicitSync_wrapper(void* d_in, void* d_out, const UNet3DParams& p, const hipStream_t stream = 0, const int deviceId = 0)
{
    hipSetDevice(deviceId);
    // for Gaussian patching & accumulating
    dim3 dimBlock_patch(p.roi_dhw, 1, 1);
    dim3 dimGrid_patch(p.roi_dhw, p.roi_dhw, 1);
    // repackaging new UNet3DParams
    UNet3DParams u3p;
    u3p.image_d = p.image_d;
    u3p.image_h = p.image_h;
    u3p.image_w = p.image_w;
    u3p.image_size = p.image_size;
    auto roi_size = p.roi_size;
    auto out_ch = p.out_ch;
    for (int slice_id = 0; slice_id < p.actual_num_slices; slice_id++)
    {
        u3p.slice_to_off_d[0] = p.slice_to_off_d[slice_id];
        u3p.slice_to_off_h[0] = p.slice_to_off_h[slice_id];
        u3p.slice_to_off_w[0] = p.slice_to_off_w[slice_id];
        u3p.patches[0] = p.patches[slice_id];
        UNet3DKiTS19PatchKernelNoOverlap<<<dimGrid_patch, dimBlock_patch, 0, stream>>>(
            &(static_cast<__half*>(d_in)[slice_id * roi_size * out_ch]), 
            static_cast<__half*>(d_out), 
            u3p);
    }
}

void UNet3DKiTS19PatchKernel_wrapper(void* d_in, void* d_out, const UNet3DParams& p, const hipStream_t stream = 0, const int deviceId = 0, const bool useCGImpl = false)
{
    if (useCGImpl)
    {
        UNet3DKiTS19PatchKernelOverlapCG_wrapper(d_in, d_out, p, stream, deviceId);
    }
    else
    {
        UNet3DKiTS19PatchKernelOverlapImplicitSync_wrapper(d_in, d_out, p, stream, deviceId);
    }
}

void UNet3DKiTS19ArgMaxKernel_wrapper(
    void* d_in, void* d_out, const UNet3DParams& p, const hipStream_t stream = 0, const int deviceId = 0)
{
    hipSetDevice(deviceId);
    // for final ArgMax
    dim3 dimBlock_argmax(p.image_w, 1, 1);
    dim3 dimGrid_argmax(p.image_d, p.image_h, 1);
    UNet3DKiTS19ArgMaxKernel<<<dimGrid_argmax, dimBlock_argmax, 0, stream>>>(
        static_cast<__half*>(d_in), static_cast<int8_t*>(d_out), p);
}

} // namespace lwis
