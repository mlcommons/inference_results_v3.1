#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2023, NVIDIA CORPORATION. All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <assert.h>
#include <stdio.h>

#include <algorithm>

#include "conv3d_3x3x3_c1_k32.h"

#include <mma.h>

using namespace nvcuda;

// <type, BLOCK_ROW_WARPS, BLOCK_COL_WARPS, WARP_ROW_TILES, WARP_COL_TILES>
// WARP_ROW_TILES is determined as `c / 16`, where c is input feature maps, BLOCK_ROW_WARPS must be 1

///////////////////////////////////////////////////////////////////////////////////////////////////

template <int ELEMENTS_PER_WARP_LOAD>
using Copy_int8_t = typename std::conditional<ELEMENTS_PER_WARP_LOAD == 32, int8_t,
    typename std::conditional<ELEMENTS_PER_WARP_LOAD == 64, uint16_t,
        typename std::conditional<ELEMENTS_PER_WARP_LOAD == 128, int,
            typename std::conditional<ELEMENTS_PER_WARP_LOAD == 256, int2, int4>::type>::type>::type>::type;

template <typename T, int ELEMENTS_PER_WARP_LOAD>
using Copy_t = Copy_int8_t<sizeof(T) / sizeof(int8_t) * ELEMENTS_PER_WARP_LOAD>;

template <int ELEMENTS_PER_THREAD>
using copy_int8_t = Copy_t<int8_t, kernel_params_int8::WARP_SIZE * ELEMENTS_PER_THREAD>;

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename T, int ELEMENTS_PER_THREAD>
union Access_t {
    Copy_t<T, kernel_params_int8::WARP_SIZE * ELEMENTS_PER_THREAD> v;
    T x[ELEMENTS_PER_THREAD];
};

////////////////////////////////////////////////////////////////////////////////////////////////////

static inline __host__ __device__ int div_up(int m, int n)
{
    return (m + n - 1) / n;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename Kernel_params>
__global__ void __launch_bounds__(Kernel_params::THREADS_PER_BLOCK, 1)
    conv_3x3x3_c1_k32_linear_kernel(Conv3d3x3x3c1k32Params params)
{
    // Naive FP32 impplementation

    typedef typename Kernel_params::Input_Data_Type Input_Data_Type;
    typedef typename Kernel_params::Output_Data_Type Output_Data_Type;
    typedef float Math_Type;

    constexpr int PAD = 1;

    constexpr int FLT_T = Kernel_params::FLT_T;
    constexpr int FLT_R = Kernel_params::FLT_R;
    constexpr int FLT_S = Kernel_params::FLT_S;

    constexpr int D_PER_CTA = Kernel_params::D_PER_CTA;
    constexpr int H_PER_CTA = Kernel_params::H_PER_CTA;
    constexpr int W_PER_CTA = Kernel_params::W_PER_CTA;

    constexpr int K_PER_CTA = Kernel_params::K_PER_CTA;

    constexpr int WARP_SIZE = Kernel_params::WARP_SIZE;
    constexpr int THREADS_PER_BLOCK = Kernel_params::THREADS_PER_BLOCK;

    const int lane_id = threadIdx.x % WARP_SIZE;
    const int warp_id = threadIdx.x / WARP_SIZE;

    constexpr int NUM_WARPS = THREADS_PER_BLOCK / WARP_SIZE;

    constexpr int FLT_SIZE = FLT_T * FLT_R * FLT_S * K_PER_CTA;
    constexpr int SMEM_D_DIM = (D_PER_CTA + FLT_T - 1);
    constexpr int SMEM_H_DIM = (H_PER_CTA + FLT_R - 1);
    constexpr int SMEM_W_DIM = (W_PER_CTA + FLT_S - 1);
    __shared__ Input_Data_Type smem[SMEM_D_DIM * SMEM_H_DIM * SMEM_W_DIM];
    __shared__ Input_Data_Type smem_flt[FLT_SIZE];

    const int n = blockIdx.z;
    const int c = blockIdx.y;
    int cta_d_begin = blockIdx.x % params.cta_per_d * D_PER_CTA;
    int cta_h_begin = blockIdx.x / params.cta_per_d % params.cta_per_h * H_PER_CTA;
    int cta_w_begin = blockIdx.x / (params.cta_per_d * params.cta_per_h) * W_PER_CTA;

    int cta_o_begin = cta_d_begin;
    int cta_p_begin = cta_h_begin;
    int cta_q_begin = cta_w_begin;

    Input_Data_Type* gmem_in = reinterpret_cast<Input_Data_Type*>(params.gmem_in) + n * params.img_stride_n
        + c * params.img_stride_c + cta_d_begin * params.img_stride_d + cta_h_begin * params.img_stride_h
        + cta_w_begin * params.img_stride_w;

    Input_Data_Type* gmem_flt = reinterpret_cast<Input_Data_Type*>(params.gmem_flt);

    Output_Data_Type* gmem_out = reinterpret_cast<Output_Data_Type*>(params.gmem_out) + n * params.out_stride_n
        + cta_o_begin * params.out_stride_o + cta_p_begin * params.out_stride_p + cta_q_begin * params.out_stride_q;

    bool is_valid = true;
    // input load is not efficient, but is much smaller than the output
    for (int w_index = lane_id; w_index < SMEM_W_DIM; w_index += WARP_SIZE)
    {
        int w = cta_w_begin - PAD + w_index;
        for (int dh_index = warp_id; dh_index < SMEM_D_DIM * SMEM_H_DIM; dh_index += NUM_WARPS)
        {
            is_valid = (w >= 0) && (w < params.img_w);
            int h_index = dh_index % SMEM_H_DIM;
            int d_index = dh_index / SMEM_H_DIM;
            int d = cta_d_begin - PAD + d_index;
            int h = cta_h_begin - PAD + h_index;
            is_valid = is_valid && (d >= 0) && (d < params.img_d);
            is_valid = is_valid && (h >= 0) && (h < params.img_h);
            smem[d_index * SMEM_H_DIM * SMEM_W_DIM + h_index * SMEM_W_DIM + w_index] = (is_valid)
                ? *(reinterpret_cast<Input_Data_Type*>(params.gmem_in) + n * params.img_stride_n
                      + d * params.img_stride_d + h * params.img_stride_h + w * params.img_stride_w)
                : Input_Data_Type(0);
        }
    }

    // Assume KCTRS format (does not matter for C == 1)
    for (int i = threadIdx.x; i < FLT_SIZE; i += THREADS_PER_BLOCK)
    {
        smem_flt[i] = gmem_flt[i];
    }

    __syncthreads();

    constexpr int Q_PER_CTA = W_PER_CTA;
    const int thread_in_cta_q = threadIdx.x % Q_PER_CTA;
    const int thread_k = threadIdx.x / Q_PER_CTA;

#pragma unroll
    for (int o_index = 0; o_index < D_PER_CTA; o_index++)
    {
#pragma unroll
        for (int p_index = 0; p_index < H_PER_CTA; p_index++)
        {
#pragma unroll
            for (int k = thread_k; k < K_PER_CTA; k += THREADS_PER_BLOCK / SMEM_W_DIM)
            {
                Math_Type sum = 0.0F;
#pragma unroll
                for (int t = 0; t < FLT_T; t++)
                {
#pragma unroll
                    for (int r = 0; r < FLT_R; r++)
                    {
#pragma unroll
                        for (int s = 0; s < FLT_S; s++)
                        {

                            Input_Data_Type val = smem[(o_index + t) * SMEM_H_DIM * SMEM_W_DIM
                                + (p_index + r) * SMEM_W_DIM + thread_in_cta_q + s];

                            sum += (float) val
                                * (float) smem_flt[k * 1 * FLT_T * FLT_R * FLT_S + t * FLT_R * FLT_S + r * FLT_S + s];
                        }
                    }
                }
                gmem_out[o_index * params.out_stride_o + p_index * params.out_stride_p
                    + thread_in_cta_q * params.out_stride_q + k * params.out_stride_k]
                    = sum;
            }
        }
    }
}

// The slicing to be implemented
// extern "C" __global__ void UNet3DKiTS19SliceKernelI8Linear(
//     const int8_t* __restrict__ d_in, int8_t* __restrict__ d_out, const UNet3DParams* p)
// {
//     int d = blockIdx.x;
//     int h = blockIdx.y;
//     int w = threadIdx.x;

//     d_out[p->roi_dhw * p->roi_dhw * d + p->roi_dhw * h + w]
//         = d_in[p->image_w * p->image_h * (p->offset_d + d) + p->image_w * (p->offset_h + h) + (p->offset_w + w)];
// }

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename Kernel_params>
__global__ void __launch_bounds__(Kernel_params::THREADS_PER_BLOCK, 1)
    conv_3x3x3_c1_k32_int8_kernel(Conv3d3x3x3c1k32Params params)
{
    /*
    x - spacial dim
    y - input c dim
    z - batch dim

    TILE_X - 32 or 64
    TILE_Y - 4
    TILE_Z - 4

    K_PER_CTA = 32

    */

    typedef typename Kernel_params::Input_Data_Type Input_Data_Type;
    typedef typename Kernel_params::Output_Data_Type Output_Data_Type;
    typedef int Math_Type;

    constexpr int PAD = 1;

    constexpr int FLT_T = Kernel_params::FLT_T;
    constexpr int FLT_R = Kernel_params::FLT_R;
    constexpr int FLT_S = Kernel_params::FLT_S;

    constexpr int D_PER_CTA = Kernel_params::D_PER_CTA;
    constexpr int H_PER_CTA = Kernel_params::H_PER_CTA;
    constexpr int W_PER_CTA = Kernel_params::W_PER_CTA;

    constexpr int K_PER_CTA = Kernel_params::K_PER_CTA;

    constexpr int WARP_SIZE = Kernel_params::WARP_SIZE;
    constexpr int THREADS_PER_BLOCK = Kernel_params::THREADS_PER_BLOCK;

    constexpr int THREADS_PER_PIXEL = Kernel_params::THREADS_PER_PIXEL;
    constexpr int ELEMENTS_PER_THREAD = WARP_SIZE / THREADS_PER_PIXEL;

    const int lane_id = threadIdx.x % WARP_SIZE;
    const int warp_id = threadIdx.x / WARP_SIZE;

    constexpr int NUM_WARPS = THREADS_PER_BLOCK / WARP_SIZE;

    constexpr int FLT_S_PAD = (FLT_S + 4 - 1) / 4 * 4;
    constexpr int FLT_SIZE = FLT_T * FLT_R * FLT_S_PAD * K_PER_CTA;
    constexpr int SMEM_D_DIM = (D_PER_CTA + FLT_T - 1);
    constexpr int SMEM_H_DIM = (H_PER_CTA + FLT_R - 1);
    constexpr int SMEM_W_DIM = (W_PER_CTA + FLT_S - 1);

    constexpr int SMEM_W_STRIDE = 4;
    __shared__ Input_Data_Type smem[SMEM_D_DIM * SMEM_H_DIM * SMEM_W_DIM * SMEM_W_STRIDE];
    __shared__ Input_Data_Type smem_flt[FLT_SIZE];

    const int n = blockIdx.z;
    const int c = blockIdx.y;
    int cta_d_begin = blockIdx.x % params.cta_per_d * D_PER_CTA;
    int cta_h_begin = blockIdx.x / params.cta_per_d % params.cta_per_h * H_PER_CTA;
    int cta_w_begin = blockIdx.x / (params.cta_per_d * params.cta_per_h) * W_PER_CTA;

    int cta_o_begin = cta_d_begin;
    int cta_p_begin = cta_h_begin;
    int cta_q_begin = cta_w_begin;

    Input_Data_Type* gmem_in = reinterpret_cast<Input_Data_Type*>(params.gmem_in) + n * params.img_stride_n
        + c * params.img_stride_c + cta_d_begin * params.img_stride_d + cta_h_begin * params.img_stride_h
        + cta_w_begin * params.img_stride_w;

    Input_Data_Type* gmem_flt = reinterpret_cast<Input_Data_Type*>(params.gmem_flt);

    Output_Data_Type* gmem_out = reinterpret_cast<Output_Data_Type*>(params.gmem_out) + n * params.out_stride_n
        + cta_o_begin * params.out_stride_o + cta_p_begin * params.out_stride_p + cta_q_begin * params.out_stride_q;

    bool is_valid = true;
    // input load is not efficient, but is much smaller than the output
    for (int w_index = lane_id; w_index < SMEM_W_DIM; w_index += WARP_SIZE)
    {
        for (int dh_index = warp_id; dh_index < SMEM_D_DIM * SMEM_H_DIM; dh_index += NUM_WARPS)
        {
            // 4th element is garbage, but would be zeroed out by filter zero-padding
            for (int iw = 0; iw < SMEM_W_STRIDE - 1; iw++)
            {
                int w = cta_w_begin - PAD + w_index + iw;
                is_valid = (w >= 0) && (w < params.img_w);
                int h_index = dh_index % SMEM_H_DIM;
                int d_index = dh_index / SMEM_H_DIM;
                int d = cta_d_begin - PAD + d_index;
                int h = cta_h_begin - PAD + h_index;
                is_valid = is_valid && (d >= 0) && (d < params.img_d);
                is_valid = is_valid && (h >= 0) && (h < params.img_h);
                smem[dh_index * SMEM_W_DIM * SMEM_W_STRIDE + w_index * SMEM_W_STRIDE + iw] = (is_valid)
                    ? *(reinterpret_cast<Input_Data_Type*>(params.gmem_in) + n * params.img_stride_n
                          + d * params.img_stride_d + h * params.img_stride_h + w * params.img_stride_w)
                    : Input_Data_Type(0);
            }
        }
    }

    // populate filter (KCTRS format C==1) properly for FLT_S_PAD
    for (int i = threadIdx.x; i < FLT_SIZE; i += THREADS_PER_BLOCK)
    {
        int ktr_index = i / FLT_S_PAD;
        int s = i % FLT_S_PAD;
        is_valid = s != 3;

        // convert smem_flt index into gmem_flt index
        // technically j == round(i * 0.75) if is_valid, but below should be faster
        int g = is_valid ? s : 0;
        int j = ktr_index * FLT_S + g;

        smem_flt[i] = is_valid ? gmem_flt[j] : Input_Data_Type(0);
    }

    __syncthreads();

    // whole CTA works in Q dimension
    const int thread_k = threadIdx.x % THREADS_PER_PIXEL;
    const int thread_in_cta_q = threadIdx.x / THREADS_PER_PIXEL;

    // NOTE/FIXME: assumption is LITTLE ENDIAN when INT8 values are packed in 4 Byte chunk
    int flt_int[ELEMENTS_PER_THREAD][FLT_T][FLT_R];
    int* smem_int_flt = reinterpret_cast<int*>(smem_flt);

    // load fliter to registers
    for (int k_index = 0; k_index < ELEMENTS_PER_THREAD; k_index++)
    {
        int k = thread_k * ELEMENTS_PER_THREAD + k_index;
        for (int t = 0; t < FLT_T; t++)
        {
            for (int r = 0; r < FLT_R; r++)
            {
                flt_int[k_index][t][r] = smem_int_flt[k * 1 * FLT_T * FLT_R * FLT_S_PAD / 4 + t * FLT_R * FLT_S_PAD / 4
                    + r * FLT_S_PAD / 4];
            }
        }
    }

    Access_t<int8_t, ELEMENTS_PER_THREAD> res;
    using copy_t = copy_int8_t<ELEMENTS_PER_THREAD>;

#pragma unroll
    for (int o_index = 0; o_index < D_PER_CTA; o_index++)
    {
#pragma unroll
        for (int p_index = 0; p_index < H_PER_CTA; p_index++)
        {
#pragma unroll
            for (int k_index = 0; k_index < ELEMENTS_PER_THREAD; k_index++)
            {
                Math_Type sum = 0;
#pragma unroll
                for (int t = 0; t < FLT_T; t++)
                {
#pragma unroll
                    for (int r = 0; r < FLT_R; r++)
                    {
                        auto vals
                            = *reinterpret_cast<int*>(&smem[(o_index + t) * SMEM_H_DIM * SMEM_W_DIM * SMEM_W_STRIDE
                                + (p_index + r) * SMEM_W_DIM * SMEM_W_STRIDE + SMEM_W_STRIDE * thread_in_cta_q]);
                        sum = __dp4a(vals, flt_int[k_index][t][r], sum);
                    }
                }
                // saturate value
                float x = __int2float_rn(sum) * params.scale;
                res.x[k_index] = __float_as_int(min(max(x + 12582912.0F, 12582785.0F), 12583039.0F));
            }
            *(reinterpret_cast<copy_t*>(&gmem_out[o_index * params.out_stride_o + p_index * params.out_stride_p
                  + thread_in_cta_q * params.out_stride_q])
                + thread_k)
                = res.v;
        }
    }
}

template <typename Kernel_params>
int conv_3x3x3_c1_k32_linear(
    const Conv3d3x3x3c1k32Context& context, Conv3d3x3x3c1k32Params& params, hipStream_t stream)
{
    assert(Kernel_params::THREADS_PER_BLOCK == Kernel_params::THREADS_PER_PIXEL * Kernel_params::K_PER_CTA);

    assert(params.img_c == 1);
    assert(params.flt_k >= 32 && params.flt_k % 32 == 0);
    assert(params.img_d % Kernel_params::D_PER_CTA == 0);
    assert(params.img_h % Kernel_params::H_PER_CTA == 0);
    assert(params.img_w % Kernel_params::W_PER_CTA == 0);

    const int block_sz = Kernel_params::THREADS_PER_BLOCK;

    params.cta_per_d = div_up(params.img_d, Kernel_params::D_PER_CTA);
    params.cta_per_h = div_up(params.img_h, Kernel_params::H_PER_CTA);
    params.cta_per_w = div_up(params.img_w, Kernel_params::W_PER_CTA);

    dim3 grid = dim3(params.cta_per_d * params.cta_per_h * params.cta_per_w, params.img_c, params.img_n);
    // const int loops = div_up(div_up(params.m, block_sz), grid);

    conv_3x3x3_c1_k32_linear_kernel<Kernel_params><<<grid, block_sz, 0, stream>>>(params);

    return 0;
}

template <typename Kernel_params>
int conv_3x3x3_c1_k32_int8(const Conv3d3x3x3c1k32Context& context, Conv3d3x3x3c1k32Params& params, hipStream_t stream)
{
    assert(Kernel_params::THREADS_PER_BLOCK
        == Kernel_params::THREADS_PER_PIXEL * Kernel_params::K_PER_CTA * Kernel_params::W_PER_CTA
            / Kernel_params::WARP_SIZE);

    assert(params.img_c == 1);
    assert(params.flt_k >= 32 && params.flt_k % 32 == 0);
    assert(params.img_d % Kernel_params::D_PER_CTA == 0);
    assert(params.img_h % Kernel_params::H_PER_CTA == 0);
    assert(params.img_w % Kernel_params::W_PER_CTA == 0);

    const int block_sz = Kernel_params::THREADS_PER_BLOCK;

    params.cta_per_d = div_up(params.img_d, Kernel_params::D_PER_CTA);
    params.cta_per_h = div_up(params.img_h, Kernel_params::H_PER_CTA);
    params.cta_per_w = div_up(params.img_w, Kernel_params::W_PER_CTA);

    dim3 grid = dim3(params.cta_per_d * params.cta_per_h * params.cta_per_w, params.img_c, params.img_n);
    // const int loops = div_up(div_up(params.m, block_sz), grid);

    conv_3x3x3_c1_k32_int8_kernel<Kernel_params><<<grid, block_sz, 0, stream>>>(params);

    return 0;
}

int conv_3x3x3_c1_k32_dispatch(
    const Conv3d3x3x3c1k32Context& context, Conv3d3x3x3c1k32Params& params, hipStream_t stream)
{

    if (params.is_fp32)
    {
        conv_3x3x3_c1_k32_linear<kernel_params_fp32>(context, params, stream);
    }
    else
    {
        conv_3x3x3_c1_k32_int8<kernel_params_int8>(context, params, stream);
    }

    return 0;
}
