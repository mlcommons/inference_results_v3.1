/*
 * Copyright (c) 2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>

#include "concatNmsOutputs.h"
#include "concatNmsOutputsKernel.cuh"

#define CHECK(status)                                          \
    do                                                         \
    {                                                          \
        auto ret = (status);                                   \
        if (ret != 0)                                          \
        {                                                      \
            std::cout << "Cuda failure: " << ret << std::endl; \
            abort();                                           \
        }                                                      \
    } while (0)




using namespace nvinfer1;
using nvinfer1::plugin::RetinanetConcatNmsOutputsPlugin;
using nvinfer1::plugin::RetinanetConcatNmsOutputsPluginCreator;

PluginFieldCollection RetinanetConcatNmsOutputsPluginCreator::mFC{};
REGISTER_TENSORRT_PLUGIN(RetinanetConcatNmsOutputsPluginCreator);



RetinanetConcatNmsOutputsPlugin::RetinanetConcatNmsOutputsPlugin(const PluginFieldCollection *fc) {
}

RetinanetConcatNmsOutputsPlugin::RetinanetConcatNmsOutputsPlugin(const void* data, size_t length) {
}

const char* RetinanetConcatNmsOutputsPlugin::getPluginType() const noexcept
{
    return "RetinanetConcatNmsOutputsPlugin";
}

const char* RetinanetConcatNmsOutputsPlugin::getPluginVersion() const noexcept
{
    return "1";
}

void RetinanetConcatNmsOutputsPlugin::setPluginNamespace(const char* libNamespace) noexcept
{
    mNamespace = libNamespace;
}

const char* RetinanetConcatNmsOutputsPlugin::getPluginNamespace() const noexcept
{
    return mNamespace.c_str();
}

void RetinanetConcatNmsOutputsPlugin::destroy() noexcept
{
    delete this;
}

IPluginV2DynamicExt* RetinanetConcatNmsOutputsPlugin::clone() const noexcept
{
    size_t sz = getSerializationSize();

    char* buff = (char*) malloc(getSerializationSize());

    // serialize is an assertion sanity check because SelectPlugin is sizeless
    serialize(buff);
    RetinanetConcatNmsOutputsPlugin* ret = new RetinanetConcatNmsOutputsPlugin(buff, sz);
    free(buff);

    return ret;
}

int RetinanetConcatNmsOutputsPlugin::getNbOutputs() const noexcept
{
    return 1;
}


DimsExprs RetinanetConcatNmsOutputsPlugin::getOutputDimensions(
    int outputIndex, const DimsExprs* inputs, int nbInputs, IExprBuilder& exprBuilder) noexcept
{

    assert(outputIndex >= 0 && outputIndex < this->getNbOutputs());
    assert(nbInputs == 4);

    DimsExprs ret;
    ret.nbDims = 2;
    ret.d[0] = inputs[0].d[0];
    ret.d[1] = exprBuilder.constant(7001);  // FIXME

    return(ret);

}

bool RetinanetConcatNmsOutputsPlugin::supportsFormatCombination(
    int pos, const PluginTensorDesc* inOut, int nbInputs, int nbOutputs) noexcept
{
    if (nbInputs != 4 || nbOutputs != 1 ) {
        printf("Wrong input or output count: %d and %d\n", nbInputs, nbOutputs);
        return false;
    }

    // Input 0 and Input 3 should be in INT32 linear format. The other inputs/outputs should be in FP32 linear format.
    DataType expectedDtype{DataType::kFLOAT};
    if (pos == 0 || pos == 3)
    {
        expectedDtype = DataType::kINT32;
    }

    if (inOut[pos].type != expectedDtype && inOut[pos].format == PluginFormat::kLINEAR)
    {
        return false;
    }

    return true;
}

void RetinanetConcatNmsOutputsPlugin::configurePlugin(
    const DynamicPluginTensorDesc* in, int nbInputs, const DynamicPluginTensorDesc* out, int nbOutputs) noexcept
{
}

int RetinanetConcatNmsOutputsPlugin::initialize() noexcept
{
    return hipSuccess;
}

void RetinanetConcatNmsOutputsPlugin::terminate() noexcept {
}

size_t RetinanetConcatNmsOutputsPlugin::getWorkspaceSize(
    const PluginTensorDesc* inputs, int nbInputs, const PluginTensorDesc* outputs, int nbOutputs) const noexcept
{
    size_t size = 0;

    return size;
}

// int RetinanetConcatNmsOutputsPlugin::enqueue(int batchSize, const void* const* inputs, void** outputs, void* workspace, hipStream_t
// stream) {
int RetinanetConcatNmsOutputsPlugin::enqueue(const PluginTensorDesc* inputDesc, const PluginTensorDesc* outputDesc,
    const void* const* inputs, void* const* outputs, void* workspace, hipStream_t stream) noexcept
{

    int batchSize = inputDesc[0].dims.d[0];
    int C0 = inputDesc[1].dims.d[1];

    assert(C0==1000);

    launch_concat_nms_outputs_gpu(batchSize,C0,
        (float*) outputs[0],
        (float*) inputs[0],
        (float*) inputs[1],
        (float*) inputs[2],
        (float*) inputs[3],
        stream);

    return 0;
}

size_t RetinanetConcatNmsOutputsPlugin::getSerializationSize() const noexcept
{
    size_t sz = 0;

    return sz;
}

void RetinanetConcatNmsOutputsPlugin::serialize(void* buffer) const noexcept
{
// Use maybe_unused attribute when updating to CUDA_STANDARD C++17
#ifndef NDEBUG
    char* d = static_cast<char*>(buffer);
    auto *d_start = d;
#endif

    assert(d == d_start + getSerializationSize());
}

nvinfer1::DataType RetinanetConcatNmsOutputsPlugin::getOutputDataType(
    int index, const nvinfer1::DataType* inputTypes, int nbInputs) const noexcept
{
    return DataType::kFLOAT;
}

template <typename T>
void RetinanetConcatNmsOutputsPlugin::write(char*& buffer, const T& val) const
{
    *reinterpret_cast<T*>(buffer) = val;
    buffer += sizeof(T);
}

template <typename T>
void RetinanetConcatNmsOutputsPlugin::read(const char*& buffer, T& val) const
{
    val = *reinterpret_cast<const T*>(buffer);
    buffer += sizeof(T);
}

const char* RetinanetConcatNmsOutputsPluginCreator::getPluginName() const noexcept
{
    return "RetinanetConcatNmsOutputsPlugin";
}

const char* RetinanetConcatNmsOutputsPluginCreator::getPluginVersion() const noexcept
{
    return "1";
}

const PluginFieldCollection* RetinanetConcatNmsOutputsPluginCreator::getFieldNames() noexcept
{
    return &mFC;
}

void RetinanetConcatNmsOutputsPluginCreator::setPluginNamespace(const char* libNamespace) noexcept
{
    mNamespace = libNamespace;
}

const char* RetinanetConcatNmsOutputsPluginCreator::getPluginNamespace() const noexcept
{
    return mNamespace.c_str();
}

IPluginV2DynamicExt* RetinanetConcatNmsOutputsPluginCreator::createPlugin(const char* name, const PluginFieldCollection* fc) noexcept
{
    return new RetinanetConcatNmsOutputsPlugin(fc);
}

IPluginV2DynamicExt* RetinanetConcatNmsOutputsPluginCreator::deserializePlugin(
    const char* name, const void* serialData, size_t serialLength) noexcept
{
    return new RetinanetConcatNmsOutputsPlugin(serialData, serialLength);
}
